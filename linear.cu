#include "hip/hip_runtime.h"
/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ops.h"
#include "cnn_helper.h"

Tensor CnnModel::add_linear_layer(Tensor input, int output_channels, bool relu)
{
  assert(input.numDim == 2);
  Linear *li = new Linear(config, input, fc_part_is, output_channels, relu);
  layers.push_back(li);
  return li->output;
}

Linear::Linear(CnnConfig config, Tensor input, IndexSpaceT<2> part_is,
               int output_channels, bool _relu)
: Op(input), relu(_relu), profiling_runtime(config.profiling),
  in_channels(input.adim[0]), out_channels(output_channels), num_replica(config.num_par_n)
{
  assert(input.numDim == 2);
  Context ctx = config.lg_ctx;
  HighLevelRuntime* runtime = config.lg_hlr;

  FieldSpace fs = runtime->create_field_space(ctx);
  {
    FieldAllocator allocator = runtime->create_field_allocator(ctx, fs);
    allocator.allocate_field(sizeof(float), FID_DATA);
  }

  Rect<2, coord_t> output_rect(Point<2>(0, 0), Point<2>(out_channels-1, input.adim[1]-1));
  IndexSpaceT<2> output_is = runtime->create_index_space(ctx, output_rect);
  LogicalRegion output_lr = runtime->create_logical_region(ctx, output_is, fs);
  LogicalRegion output_grad_lr = runtime->create_logical_region(ctx, output_is, fs);
  Transform<2, 2, coord_t> transform;
  int extent_c = (out_channels + config.fc_num_par_c - 1) / config.fc_num_par_c;
  int extent_n = (input.adim[1] + config.fc_num_par_n - 1) / config.fc_num_par_n;
  Rect<2, coord_t> extent(Point<2>(0, 0), Point<2>(extent_c-1, extent_n-1));
  transform[0][0] = extent_c; transform[0][1] = 0;
  transform[1][0] = 0; transform[1][1] = extent_n;
  IndexPartition output_ip =
    runtime->create_partition_by_restriction(ctx, output_is, part_is, transform, extent);
  assert(runtime->is_index_partition_disjoint(ctx, output_ip));
  assert(runtime->is_index_partition_complete(ctx, output_ip));
  LogicalPartition output_lp = runtime->get_logical_partition(ctx, output_lr, output_ip);
  LogicalPartition output_grad_lp = runtime->get_logical_partition(ctx, output_grad_lr, output_ip);

  // Note: we only need replica's grad, so no need to create lr/lp for forward
  Rect<2, coord_t> replica_rect(Point<2>(0, 0),
                       Point<2>(in_channels*config.fc_num_par_c-1, input.adim[1]-1));
  IndexSpaceT<2> replica_is = runtime->create_index_space(ctx, replica_rect);
  LogicalRegion replica_lr = runtime->create_logical_region(ctx, replica_is, fs);
  transform[0][0] = in_channels;
  transform[1][1] = extent_n;
  Rect<2, coord_t> extent_r(Point<2>(0, 0), Point<2>(in_channels-1, extent_n-1));
  IndexPartition replica_ip =
    runtime->create_partition_by_restriction(ctx, replica_is, part_is, transform, extent_r);
  assert(runtime->is_index_partition_disjoint(ctx, replica_ip));
  assert(runtime->is_index_partition_complete(ctx, replica_ip));
  LogicalPartition replica_lp = runtime->get_logical_partition(ctx, replica_lr, replica_ip);
  TensorWithGrad replica_tensor;
  replica_tensor.region_grad = replica_lr;
  replica_tensor.partition_grad = replica_lp;
  locals[0] = replica_tensor;
  // Create subpartitions for backward prop aggregation
  for (int i = 0; i < config.fc_num_par_c; i++) {
    transform[0][0] = input.pdim[0];
    transform[1][1] = input.pdim[1];
    Rect<2, coord_t> ext(Point<2>(in_channels*i, 0),
                         Point<2>(in_channels*i+input.pdim[0]-1, input.pdim[1]));
    IndexPartition ip =
      runtime->create_partition_by_restriction(ctx, replica_is, part_is, transform, ext);
    assert(runtime->is_index_partition_disjoint(ctx, ip));
    replica_sub_lps[i] = runtime->get_logical_partition(ctx, replica_lr, ip);
  }

  Rect<2, coord_t> kernel_rect(Point<2>(0, 0), Point<2>(out_channels * in_channels-1, config.fc_num_par_n-1));
  IndexSpaceT<2> kernel_is = runtime->create_index_space(ctx, kernel_rect);
  LogicalRegion kernel_lr = runtime->create_logical_region(ctx, kernel_is, fs);
  LogicalRegion kernel_grad_lr = runtime->create_logical_region(ctx, kernel_is, fs);
  transform[0][0] = extent_c * in_channels;
  transform[1][1] = 1;
  Rect<2, coord_t> extent_k(Point<2>(0, 0), Point<2>(extent_c*in_channels-1, 0));
  printf("extent_k(%dx%d %d)\n", extent_c, in_channels, 1);
  IndexPartition kernel_ip =
    runtime->create_partition_by_restriction(ctx, kernel_is, part_is, transform, extent_k);
  assert(runtime->is_index_partition_disjoint(ctx, kernel_ip));
  assert(runtime->is_index_partition_complete(ctx, kernel_ip));
  LogicalPartition kernel_lp = runtime->get_logical_partition(ctx, kernel_lr, kernel_ip);
  LogicalPartition kernel_grad_lp = runtime->get_logical_partition(ctx, kernel_grad_lr, kernel_ip);
  TensorWithGrad kernel_tensor;
  kernel_tensor.region = kernel_lr;
  kernel_tensor.partition = kernel_lp;
  kernel_tensor.region_grad = kernel_grad_lr;
  kernel_tensor.partition_grad = kernel_grad_lp;
  locals[1] = kernel_tensor;

  Rect<2, coord_t> bias_rect(Point<2>(0, 0), Point<2>(out_channels-1, config.fc_num_par_n-1));
  IndexSpaceT<2> bias_is = runtime->create_index_space(ctx, bias_rect);
  LogicalRegion bias_lr = runtime->create_logical_region(ctx, bias_is, fs);
  LogicalRegion bias_grad_lr = runtime->create_logical_region(ctx, bias_is, fs);
  transform[0][0] = extent_c;
  transform[1][1] = 1;
  Rect<2, coord_t> extent_b(Point<2>(0, 0), Point<2>(extent_c-1,0));
  IndexPartition bias_ip =
    runtime->create_partition_by_restriction(ctx, bias_is, part_is, transform, extent_b);
  assert(runtime->is_index_partition_disjoint(ctx, bias_ip));
  assert(runtime->is_index_partition_complete(ctx, bias_ip));
  LogicalPartition bias_lp = runtime->get_logical_partition(ctx, bias_lr, bias_ip);
  LogicalPartition bias_grad_lp = runtime->get_logical_partition(ctx, bias_grad_lr, bias_ip);
  TensorWithGrad bias_tensor;
  bias_tensor.region = bias_lr;
  bias_tensor.partition = bias_lp;
  bias_tensor.region_grad = bias_grad_lr;
  bias_tensor.partition_grad = bias_grad_lp;
  locals[2] = bias_tensor;

  output.numDim = 2;
  output.adim[0] = out_channels;
  output.adim[1] = input.adim[1];
  output.pdim[0] = extent_c;
  output.pdim[1] = extent_n;
  output.region = output_lr;
  output.partition = output_lp;
  output.region_grad = output_grad_lr;
  output.partition_grad = output_grad_lp;

  // Every partition reads all in_channels
  transform[0][0] = 0;
  transform[1][1] = extent_n;
  Rect<2, coord_t> extent_i(Point<2>(0, 0), Point<2>(in_channels-1, extent_n-1));
  IndexSpaceT<2> input_is = IndexSpaceT<2>(inputs[0].region.get_index_space());
  IndexPartition input_ip 
     = runtime->create_partition_by_restriction(ctx, input_is, part_is, transform, extent_i);
  input_lps[0] = runtime->get_logical_partition(ctx, inputs[0].region, input_ip);
}

/*
  regions[0]: input
  regions[1]: output
  regions[2]: replica
  regions[3]: kernel
  regions[4]: bias
*/
OpMeta* Linear::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  const int BLKSIZE = 512;
  assert(regions.size() == 5);
  assert(task->regions.size() == 5);
  const Linear* linear = (Linear*) task->args;
  CnnHandle handle = *((const CnnHandle*) task->local_args);
  const AccessorRO<float, 2> acc_input(regions[0], FID_DATA);
  const AccessorRO<float, 2> acc_output(regions[1], FID_DATA);
  const AccessorWO<float, 2> acc_kernel(regions[3], FID_DATA);
  const AccessorWO<float, 2> acc_bias(regions[4], FID_DATA);
  Rect<2> rect_input, rect_output, rect_replica, rect_kernel, rect_bias;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_replica = runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_kernel = runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(ctx, task->regions[4].region.get_index_space());
  assert(rect_replica.volume() == rect_input.volume());
  assert(acc_kernel.accessor.is_dense_arbitrary(rect_kernel));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  float* kernel_ptr = acc_kernel.ptr(rect_kernel.lo);
  float* bias_ptr = acc_bias.ptr(rect_bias.lo);
  hiprandGenerator_t genGPU;
  hiprandCreateGenerator(&genGPU, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(genGPU, 1234ULL);
  int input_channels = rect_input.hi[0] - rect_input.lo[0] + 1;
  int output_channels = rect_output.hi[0] - rect_output.lo[0] + 1;
  int batch_size = linear->output.pdim[1];
  printf("init linear (input): in_c(%d) out_c(%d) batch_size(%d)\n", input_channels, output_channels, batch_size);
  LinearMeta* m = new LinearMeta(handle);
#ifndef DISABLE_COMPUTATION
  m->relu = linear->relu;
  m->input_channels = input_channels;
  m->output_channels = output_channels;
  m->batch_size = batch_size;

  coord_t kernel_elements = input_channels * linear->output.pdim[0];
  float factor = 1.0f / sqrt(input_channels);
  assert(kernel_elements == rect_kernel.volume());
  hiprandGenerateUniform(genGPU, kernel_ptr, kernel_elements);
  int num_blocks = (kernel_elements + BLKSIZE - 1) / BLKSIZE;
  scale_kernel<<<num_blocks, BLKSIZE>>>(kernel_ptr, kernel_elements, -factor, factor);
  hiprandGenerateUniform(genGPU, bias_ptr, linear->output.pdim[0]);
  num_blocks = (linear->output.pdim[0] + BLKSIZE - 1) / BLKSIZE;
  scale_kernel<<<num_blocks, BLKSIZE>>>(bias_ptr, linear->output.pdim[0], -factor, factor);
  hiprandDestroyGenerator(genGPU);

  float* dram_one_ptr = (float *) malloc(sizeof(float) * batch_size);
  for (int i = 0; i < batch_size; i++)
    dram_one_ptr[i] = 1.0f;
  checkCUDA(hipMalloc(&m->one_ptr, sizeof(float) * batch_size));
  checkCUDA(hipMemcpy(m->one_ptr, dram_one_ptr,
                       sizeof(float) * batch_size, hipMemcpyDeviceToDevice));
  if (m->relu) {
    checkCUDNN(hipdnnCreateActivationDescriptor(&m->actiDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, HIPDNN_ACTIVATION_RELU,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    checkCUDNN(hipdnnCreateTensorDescriptor(&m->outputTensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          batch_size, output_channels, 1, 1));
  }
#endif
  return m;
}

void Linear::init(const CnnModel& model)
{
  ArgumentMap argmap;
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, model.fc_part_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    CnnHandle handle = model.cnn_handlers[idx++];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(CnnHandle)));
  }
  IndexLauncher init_launcher(LINEAR_INIT_TASK_ID, model.fc_part_is,
                              TaskArgument(this, sizeof(Linear)), argmap);
  init_launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  init_launcher.add_field(0, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(output.partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  init_launcher.add_field(1, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[0].partition_grad, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, locals[0].region_grad));
  init_launcher.add_field(2, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[1].partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, locals[1].region));
  init_launcher.add_field(3, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[2].partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, locals[2].region));
  init_launcher.add_field(4, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, init_launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*
  regions[0](I); input
  regions[1](O): output
  regions[2](I): kernel
  regions[3](I): bias
*/
__host__
void Linear::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
#ifndef DISABLE_COMPUTATION
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  float alpha = 1.0f, beta = 0.0f;
  const Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  int input_channels = m->input_channels;
  int output_channels = m->output_channels;
  int batch_size = m->batch_size;
  const float *one_ptr = m->one_ptr;
  const AccessorRO<float, 2> acc_input(regions[0], FID_DATA);
  const AccessorWO<float, 2> acc_output(regions[1], FID_DATA);
  const AccessorRO<float, 2> acc_kernel(regions[2], FID_DATA);
  const AccessorRO<float, 2> acc_bias(regions[3], FID_DATA);
  Rect<2> rect_input, rect_output, rect_kernel, rect_bias;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_kernel = runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  // make sure the sizes match
  assert(rect_input.volume() == input_channels * batch_size);
  assert(rect_output.volume() == output_channels * batch_size);
  assert(rect_kernel.volume() == input_channels * output_channels);
  assert(rect_bias.volume() == output_channels);
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  //assert(acc_replica.accessor.is_dense_arbitrary(rect_replica));
  assert(acc_kernel.accessor.is_dense_arbitrary(rect_kernel));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *output_ptr = acc_output.ptr(rect_output.lo);
  //float *replica_ptr = acc_output.ptr(rect_replica.lo);
  const float *kernel_ptr = acc_kernel.ptr(rect_kernel.lo);
  const float *bias_ptr = acc_bias.ptr(rect_bias.lo);
  //float *pre_relu_ptr = (m->relu) ? m->pre_relu : output_ptr;

  //checkCUDA(hipMemcpy(replica_ptr, input_ptr, rect_input.volume() * sizeof(float),
  //                     hipMemcpyDeviceToDevice));
  hipEvent_t t_start, t_end;
  if (linear->profiling_runtime) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        output_channels, batch_size, input_channels,
                        &alpha, kernel_ptr, input_channels,
                        input_ptr, input_channels, &beta,
                        output_ptr, output_channels));
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        output_channels, batch_size, 1,
                        &alpha, bias_ptr, 1,
                        one_ptr, 1, &alpha,
                        output_ptr, output_channels));
  if (m->relu) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
                                      &alpha, m->outputTensor, output_ptr,
                                      &beta, m->outputTensor, output_ptr));
  }
  if (linear->profiling_runtime) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Linear forward time = %.2lfms\n", elapsed);
  }
#endif
}

void Linear::forward(const CnnModel& model)
{
  ArgumentMap argmap;
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, model.fc_part_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(LINEAR_FWD_TASK_ID, model.fc_part_is,
                         TaskArgument(this, sizeof(Linear)), argmap);
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(output.partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  launcher.add_field(1, FID_DATA);
  //launcher.add_region_requirement(
  //    RegionRequirement(locals[0].partition, 0/*projection id*/,
  //                      WRITE_DISCARD, EXCLUSIVE, locals[0].region));
  //launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(locals[1].partition, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[1].region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(locals[2].partition, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[2].region));
  launcher.add_field(3, FID_DATA);

  runtime->execute_index_space(ctx, launcher);
}

/*
  regions[0](I): input
  regions[1](O): replica_grad
  regions[2](I): output
  regions[3](I/O): output_grad
  regions[4](I): filter
  regions[5](O): filter_grad
  regions[6](O): bias_grad
*/
__host__
void Linear::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
#ifndef DISABLE_COMPUTATION
  assert(regions.size() == 7);
  assert(task->regions.size() == 7);
  float alpha = 1.0f, beta = 0.0f;
  const Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  int input_channels = m->input_channels;
  int output_channels = m->output_channels;
  int batch_size = m->batch_size;
  const float *one_ptr = m->one_ptr;
  const AccessorRO<float, 2> acc_input(regions[0], FID_DATA);
  const AccessorWO<float, 2> acc_replica_grad(regions[1], FID_DATA);
  const AccessorRO<float, 2> acc_output(regions[2], FID_DATA);
  const AccessorRW<float, 2> acc_output_grad(regions[3], FID_DATA);
  const AccessorRO<float, 2> acc_kernel(regions[4], FID_DATA);
  const AccessorWO<float, 2> acc_kernel_grad(regions[5], FID_DATA);
  const AccessorWO<float, 2> acc_bias_grad(regions[6], FID_DATA);
  Rect<2> rect_input, rect_replica_grad, rect_output, rect_output_grad,
          rect_kernel, rect_kernel_grad, rect_bias_grad;
  rect_input =
    runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_replica_grad =
    runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_output =
    runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_output_grad =
    runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  rect_kernel =
    runtime->get_index_space_domain(ctx, task->regions[4].region.get_index_space());
  rect_kernel_grad =
    runtime->get_index_space_domain(ctx, task->regions[5].region.get_index_space());
  rect_bias_grad =
    runtime->get_index_space_domain(ctx, task->regions[6].region.get_index_space());
  // make sure the sizes match
  assert(rect_input.volume() == input_channels * batch_size);
  assert(rect_replica_grad.volume() == input_channels * batch_size);
  assert(rect_output.volume() == output_channels * batch_size);
  assert(rect_output_grad.volume() == output_channels * batch_size);
  assert(rect_kernel.volume() == input_channels * output_channels);
  assert(rect_kernel_grad.volume() == input_channels * output_channels);
  assert(rect_bias_grad.volume() == output_channels);
  // make sure all regions are dense
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_replica_grad.accessor.is_dense_arbitrary(rect_replica_grad));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  assert(acc_output_grad.accessor.is_dense_arbitrary(rect_output_grad));
  assert(acc_kernel.accessor.is_dense_arbitrary(rect_kernel));
  assert(acc_kernel_grad.accessor.is_dense_arbitrary(rect_kernel_grad));
  assert(acc_bias_grad.accessor.is_dense_arbitrary(rect_bias_grad));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *replica_grad_ptr = acc_replica_grad.ptr(rect_replica_grad.lo);
  const float *output_ptr = acc_output.ptr(rect_output.lo);
  float *output_grad_ptr = acc_output_grad.ptr(rect_output_grad.lo);
  const float *kernel_ptr = acc_kernel.ptr(rect_kernel.lo);
  float *kernel_grad_ptr = acc_kernel_grad.ptr(rect_kernel_grad.lo);
  float *bias_grad_ptr = acc_bias_grad.ptr(rect_bias_grad.lo);

  hipEvent_t t_start, t_end;
  if (linear->profiling_runtime) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
  if (m->relu) {
    int n = rect_output.volume();
    reluBackward<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(output_grad_ptr, output_ptr, n);
  }

  // Compute weight gradiant
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        input_channels, output_channels, batch_size,
                        &alpha, input_ptr, input_channels,
                        output_grad_ptr, output_channels,
                        &beta, kernel_grad_ptr, input_channels));
  // Compute bias gradiant
  checkCUDA(hipblasSgemv(m->handle.blas, HIPBLAS_OP_N,
                        output_channels, batch_size,
                        &alpha, output_grad_ptr, output_channels,
                        one_ptr, 1,
                        &beta, bias_grad_ptr, 1));
  // Compute data gradiant
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        input_channels, batch_size, output_channels,
                        &alpha, kernel_ptr, input_channels,
                        output_grad_ptr, output_channels,
                        &beta, replica_grad_ptr, input_channels));
  if (linear->profiling_runtime) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Linear backward time = %.2lfms\n", elapsed);
  }
#endif
}

/*
  regions[0](O): input_grad
  regions[1..fc_num_par_c]: subreplicas
*/
__host__
void Linear::backward2_task(const Task *task,
                            const std::vector<PhysicalRegion> &regions,
                            Context ctx, Runtime *runtime)
{
#ifndef DISABLE_COMPUTATION
  float alpha = 1.0f;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  const AccessorWO<float, 2> acc_input(regions[0], FID_DATA);
  Rect<2> rect_input, rect_replica;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  float *input_ptr = acc_input.ptr(rect_input.lo);
  for (int i = 1; i < task->regions.size(); i++) {
    const AccessorRO<float, 2> acc_replica(regions[i], FID_DATA);
    rect_replica = runtime->get_index_space_domain(ctx, task->regions[i].region.get_index_space());
    //printf("rect_replica.hi = %lld lo = %lld\n", rect_replica.hi[0], rect_replica.lo[0]);
    //printf("rect_replica.hi = %lld lo = %lld\n", rect_replica.hi[1], rect_replica.lo[1]);
    //printf("rect_input.hi = %lld lo = %lld\n", rect_input.hi[0], rect_input.lo[0]);
    //printf("rect_input.hi = %lld lo = %lld\n", rect_input.hi[1], rect_input.lo[1]);
    assert(rect_replica.volume() == rect_input.volume());
    assert(acc_replica.accessor.is_dense_arbitrary(rect_replica));
    const float *replica_ptr = acc_replica.ptr(rect_replica.lo);
    if (i == 1)
      checkCUDA(hipblasScopy(m->handle.blas, rect_input.volume(),
                            replica_ptr, 1, input_ptr, 1));
    else
      checkCUDA(hipblasSaxpy(m->handle.blas, rect_input.volume(),
                            &alpha, replica_ptr, 1, input_ptr, 1));
  }
#endif
}

void Linear::backward(const CnnModel& model)
{
  ArgumentMap argmap;
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, model.fc_part_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  {
    IndexLauncher launcher(LINEAR_BWD_TASK_ID, model.fc_part_is,
                           TaskArgument(this, sizeof(Linear)), argmap);
    // regions[0](I): input
    launcher.add_region_requirement(
        RegionRequirement(input_lps[0], 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, inputs[0].region));
    launcher.add_field(0, FID_DATA);
    // regions[1](O): replica_grad (we only need grad tensors)
    launcher.add_region_requirement(
        RegionRequirement(locals[0].partition_grad, 0/*projection id*/,
                          WRITE_DISCARD, EXCLUSIVE, locals[0].region_grad));
    launcher.add_field(1, FID_DATA);
    // regions[2](I): output
    launcher.add_region_requirement(
        RegionRequirement(output.partition, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, output.region));
    launcher.add_field(2, FID_DATA);
    // regions[3](I/O): output_grad
    launcher.add_region_requirement(
        RegionRequirement(output.partition_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, output.region_grad));
    launcher.add_field(3, FID_DATA);
    // regions[4](I): filter
    launcher.add_region_requirement(
        RegionRequirement(locals[1].partition, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, locals[1].region));
    launcher.add_field(4, FID_DATA);
    // regions[5](O): filter_grad
    launcher.add_region_requirement(
        RegionRequirement(locals[1].partition_grad, 0/*projection id*/,
                          WRITE_DISCARD, EXCLUSIVE, locals[1].region_grad));
    launcher.add_field(5, FID_DATA);
    // regions[6](O): bias_grad
    launcher.add_region_requirement(
        RegionRequirement(locals[2].partition_grad, 0/*projection id*/,
                          WRITE_DISCARD, EXCLUSIVE, locals[2].region_grad));
    launcher.add_field(6, FID_DATA);
    runtime->execute_index_space(ctx, launcher);
  }
  {
    // We aggregate parameters from replica tensor to input tensor
    IndexLauncher launcher2(LINEAR_BWD2_TASK_ID, model.fc_part_is,
                            TaskArgument(this, sizeof(Linear)), argmap);
    launcher2.add_region_requirement(
        RegionRequirement(inputs[0].partition_grad, 0/*projection id*/,
                          WRITE_DISCARD, EXCLUSIVE, inputs[0].region_grad));
    launcher2.add_field(0, FID_DATA);
    for (int i = 0; i < model.config.fc_num_par_c; i++) {
      launcher2.add_region_requirement(
          RegionRequirement(replica_sub_lps[i], 0/*partition id*/,
                            READ_ONLY, EXCLUSIVE, locals[0].region_grad));
      launcher2.add_field(i + 1, FID_DATA);
    }
    runtime->execute_index_space(ctx, launcher2);
  }
}

/*
  regions[0](I/O): filter_grad
  regions[1](I/O): bias_grad
*/
__host__
void Linear::update_task(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const Linear* linear = (Linear*) task->args;
  const AccessorRW<float, 2> acc_filter(regions[0], FID_DATA);
  const AccessorRW<float, 2> acc_bias(regions[1], FID_DATA);
  Rect<2> rect_filter, rect_bias;
  rect_filter =
    runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_bias =
    runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  size_t filter_size = rect_filter.volume() / linear->num_replica;
  size_t bias_size = rect_bias.volume() / linear->num_replica;
  assert(filter_size == linear->in_channels * linear->out_channels);
  assert(bias_size == linear->out_channels);
  assert(acc_filter.accessor.is_dense_arbitrary(rect_filter));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  float *filter_ptr = acc_filter.ptr(rect_filter.lo);
  float *bias_ptr = acc_bias.ptr(rect_bias.lo);
  updateGAS(filter_ptr, filter_size, linear->num_replica);
  updateGAS(bias_ptr, bias_size, linear->num_replica);
}

__host__
void Linear::update(const CnnModel& model)
{
  assert(num_replica > 0);
  // Only aggregate parameters if more than one replica
  if (num_replica > 1) {
    Context ctx = model.config.lg_ctx;
    Runtime* runtime = model.config.lg_hlr;
    TaskLauncher launcher(LINEAR_UPD_TASK_ID, TaskArgument(this, sizeof(Linear)));
    launcher.add_region_requirement(
      RegionRequirement(locals[1].region, READ_WRITE, EXCLUSIVE, locals[1].region));
    launcher.add_field(0, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(locals[2].region, READ_WRITE, EXCLUSIVE, locals[2].region));
    launcher.add_field(1, FID_DATA);
    runtime->execute_task(ctx, launcher);
  }
}
