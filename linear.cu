#include "hip/hip_runtime.h"
/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ops.h"
#include "cnn_helper.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hiprand.h>

Tensor CnnModel::add_linear_layer(Tensor input, int output_channels, bool relu)
{
  assert(input.numDim == 2);
  Linear *li = new Linear(config, input, fc_part_is, output_channels, relu);
  layers.push_back(li);
  return li->output;
}

Linear::Linear(CnnConfig config, Tensor input, IndexSpaceT<2> part_is,
               int output_channels, bool _relu)
: Op(input), relu(_relu)
{
  assert(input.numDim == 2);
  Context ctx = config.lg_ctx;
  HighLevelRuntime* runtime = config.lg_hlr;

  FieldSpace fs = runtime->create_field_space(ctx);
  {
    FieldAllocator allocator = runtime->create_field_allocator(ctx, fs);
    allocator.allocate_field(sizeof(float), FID_DATA);
  }

  Rect<2, coord_t> output_rect(Point<2>(0, 0), Point<2>(output_channels-1, input.adim[1]-1));
  IndexSpaceT<2> output_is = runtime->create_index_space(ctx, output_rect);
  LogicalRegion output_lr = runtime->create_logical_region(ctx, output_is, fs);
  Transform<2, 2, coord_t> transform;
  int extent_c = (output_channels + config.fc_num_par_c - 1) / config.fc_num_par_c;
  int extent_n = (input.adim[1] + config.fc_num_par_n - 1) / config.fc_num_par_n;
  int input_channels = input.adim[0];
  Rect<2, coord_t> extent(Point<2>(0, 0), Point<2>(extent_c-1, extent_n-1));
  transform[0][0] = extent_c; transform[0][1] = 0;
  transform[1][0] = 0; transform[1][1] = extent_n;
  IndexPartition output_ip =
    runtime->create_partition_by_restriction(ctx, output_is, part_is, transform, extent);
  LogicalPartition output_lp = runtime->get_logical_partition(ctx, output_lr, output_ip);
  
  Rect<2, coord_t> replica_rect(Point<2>(0, 0),
                       Point<2>(input_channels*config.fc_num_par_c-1, input.adim[1]-1));
  IndexSpaceT<2> replica_is = runtime->create_index_space(ctx, replica_rect);
  LogicalRegion replica_lr = runtime->create_logical_region(ctx, replica_is, fs);
  transform[0][0] = input_channels;
  transform[1][1] = extent_n;
  Rect<2, coord_t> extent_r(Point<2>(0, 0), Point<2>(input_channels-1, extent_n-1));
  IndexPartition replica_ip =
    runtime->create_partition_by_restriction(ctx, replica_is, part_is, transform, extent_r);
  LogicalPartition replica_lp = runtime->get_logical_partition(ctx, replica_lr, replica_ip);
  TensorWithGrad replica_tensor;
  replica_tensor.region = replica_lr;
  replica_tensor.partition = replica_lp;
  locals[0] = replica_tensor;
  // Create subpartitions for backward prop aggregation
  for (int i = 0; i < config.fc_num_par_c; i++) {
    transform[0][0] = input.pdim[0];
    transform[1][1] = input.pdim[1];
    Rect<2, coord_t> ext(Point<2>(input_channels*i, 0),
                         Point<2>(input_channels*i+input.pdim[0]-1, input.pdim[1]));
    IndexPartition ip =
      runtime->create_partition_by_restriction(ctx, replica_is, part_is, transform, ext);
    replica_sub_lps[i] = runtime->get_logical_partition(ctx, replica_lr, ip);
  }

  Rect<2, coord_t> kernel_rect(Point<2>(0, 0), Point<2>(output_channels * input_channels-1, config.fc_num_par_n-1));
  IndexSpaceT<2> kernel_is = runtime->create_index_space(ctx, kernel_rect);
  LogicalRegion kernel_lr = runtime->create_logical_region(ctx, kernel_is, fs);
  transform[0][0] = extent_c * input_channels;
  transform[1][1] = 1;
  Rect<2, coord_t> extent_k(Point<2>(0, 0), Point<2>(extent_c*input_channels-1, 0));
  printf("extent_k(%dx%d %d)\n", extent_c, input_channels, 1);
  IndexPartition kernel_ip =
    runtime->create_partition_by_restriction(ctx, kernel_is, part_is, transform, extent_k);
  LogicalPartition kernel_lp = runtime->get_logical_partition(ctx, kernel_lr, kernel_ip);
  TensorWithGrad kernel_tensor;
  kernel_tensor.region = kernel_lr;
  kernel_tensor.partition = kernel_lp;
  locals[1] = kernel_tensor;

  Rect<2, coord_t> bias_rect(Point<2>(0, 0), Point<2>(output_channels-1, config.fc_num_par_n-1));
  IndexSpaceT<2> bias_is = runtime->create_index_space(ctx, bias_rect);
  LogicalRegion bias_lr = runtime->create_logical_region(ctx, bias_is, fs);
  transform[0][0] = extent_c;
  transform[1][1] = 1;
  Rect<2, coord_t> extent_b(Point<2>(0, 0), Point<2>(extent_c-1,0));
  IndexPartition bias_ip =
    runtime->create_partition_by_restriction(ctx, bias_is, part_is, transform, extent_b);
  LogicalPartition bias_lp = runtime->get_logical_partition(ctx, bias_lr, bias_ip);
  TensorWithGrad bias_tensor;
  bias_tensor.region = bias_lr;
  bias_tensor.partition = bias_lp;
  locals[2] = bias_tensor;

  output.numDim = 2;
  output.adim[0] = output_channels;
  output.adim[1] = input.adim[1];
  output.pdim[0] = extent_c;
  output.pdim[1] = extent_n;
  output.region = output_lr;
  output.partition = output_lp;

  // Every partition reads all input_channels
  transform[0][0] = 0;
  transform[1][1] = extent_n;
  Rect<2, coord_t> extent_i(Point<2>(0, 0), Point<2>(input_channels-1, extent_n-1));
  IndexSpaceT<2> input_is = IndexSpaceT<2>(inputs[0].region.get_index_space());
  IndexPartition input_ip 
     = runtime->create_partition_by_restriction(ctx, input_is, part_is, transform, extent_i);
  input_lps[0] = runtime->get_logical_partition(ctx, inputs[0].region, input_ip);
}

/*
  regions[0]: input
  regions[1]: output
  regions[2]: replica
  regions[3]: kernel
  regions[4]: bias
*/
OpMeta* Linear::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  const int BLKSIZE = 512;
  assert(regions.size() == 5);
  assert(task->regions.size() == 5);
  const Linear* linear = (Linear*) task->args;
  CnnHandle handle = *((const CnnHandle*) task->local_args);
  const AccessorRO<float, 2> acc_input(regions[0], FID_DATA);
  const AccessorRO<float, 2> acc_output(regions[1], FID_DATA);
  const AccessorWO<float, 2> acc_kernel(regions[3], FID_DATA);
  const AccessorWO<float, 2> acc_bias(regions[4], FID_DATA);
  Rect<2> rect_input, rect_output, rect_replica, rect_kernel, rect_bias;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_replica = runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_kernel = runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(ctx, task->regions[4].region.get_index_space());
  assert(rect_replica.volume() == rect_input.volume());
  assert(acc_kernel.accessor.is_dense_arbitrary(rect_kernel));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  float* kernel_ptr = acc_kernel.ptr(rect_kernel.lo);
  float* bias_ptr = acc_bias.ptr(rect_bias.lo);
  hiprandGenerator_t genGPU;
  hiprandCreateGenerator(&genGPU, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(genGPU, 1234ULL);
  int input_channels = rect_input.hi[0] - rect_input.lo[0] + 1;
  int output_channels = rect_output.hi[0] - rect_output.lo[0] + 1;
  int batch_size = linear->output.pdim[1];
  printf("init linear (input): in_c(%d) out_c(%d) batch_size(%d)\n", input_channels, output_channels, batch_size);

  coord_t kernel_elements = input_channels * linear->output.pdim[0];
  float factor = 1.0f / sqrt(input_channels);
  assert(kernel_elements == rect_kernel.volume());
  hiprandGenerateUniform(genGPU, kernel_ptr, kernel_elements);
  int num_blocks = (kernel_elements + BLKSIZE - 1) / BLKSIZE;
  scale_kernel<<<num_blocks, BLKSIZE>>>(kernel_ptr, kernel_elements, -factor, factor);
  hiprandGenerateUniform(genGPU, bias_ptr, linear->output.pdim[0]);
  num_blocks = (linear->output.pdim[0] + BLKSIZE - 1) / BLKSIZE;
  scale_kernel<<<num_blocks, BLKSIZE>>>(bias_ptr, linear->output.pdim[0], -factor, factor);
  hiprandDestroyGenerator(genGPU);

  LinearMeta* m = new LinearMeta(handle);
  m->relu = linear->relu;
  m->input_channels = input_channels;
  m->output_channels = output_channels;
  m->batch_size = batch_size;
  float* dram_one_ptr = (float *) malloc(sizeof(float) * batch_size);
  for (int i = 0; i < batch_size; i++)
    dram_one_ptr[i] = 1.0f;
  checkCUDA(hipMalloc(&m->one_ptr, sizeof(float) * batch_size));
  checkCUDA(hipMemcpy(m->one_ptr, dram_one_ptr,
                       sizeof(float) * batch_size, hipMemcpyDeviceToDevice));
  if (m->relu) {
    checkCUDA(hipMalloc(&m->pre_relu, rect_output.volume() * sizeof(float)));
    checkCUDNN(hipdnnCreateActivationDescriptor(&m->actiDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, HIPDNN_ACTIVATION_RELU,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    checkCUDNN(hipdnnCreateTensorDescriptor(&m->outputTensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          batch_size, output_channels, 1, 1));
  }
  return m;
}

void Linear::init(const CnnModel& model)
{
  ArgumentMap argmap;
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, model.fc_part_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    CnnHandle handle = model.cnn_handlers[idx++];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(CnnHandle)));
  }
  IndexLauncher init_launcher(LINEAR_INIT_TASK_ID, model.fc_part_is,
                              TaskArgument(this, sizeof(Linear)), argmap);
  init_launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  init_launcher.add_field(0, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(output.partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  init_launcher.add_field(1, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[0].partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, locals[0].region));
  init_launcher.add_field(2, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[1].partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, locals[1].region));
  init_launcher.add_field(3, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[2].partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, locals[2].region));
  init_launcher.add_field(4, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, init_launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*
  regions[0](I); input
  regions[1](O): output
  regions[2](I): kernel
  regions[3](I): bias
*/
__host__
void Linear::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  float alpha = 1.0f, beta = 0.0f;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  int input_channels = m->input_channels;
  int output_channels = m->output_channels;
  int batch_size = m->batch_size;
  const float *one_ptr = m->one_ptr;
  const AccessorRO<float, 2> acc_input(regions[0], FID_DATA);
  const AccessorWO<float, 2> acc_output(regions[1], FID_DATA);
  const AccessorRO<float, 2> acc_kernel(regions[2], FID_DATA);
  const AccessorRO<float, 2> acc_bias(regions[3], FID_DATA);
  Rect<2> rect_input, rect_output, rect_kernel, rect_bias;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_kernel = runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  // make sure the sizes match
  assert(rect_input.volume() == input_channels * batch_size);
  assert(rect_output.volume() == output_channels * batch_size);
  assert(rect_kernel.volume() == input_channels * output_channels);
  assert(rect_bias.volume() == output_channels);
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  //assert(acc_replica.accessor.is_dense_arbitrary(rect_replica));
  assert(acc_kernel.accessor.is_dense_arbitrary(rect_kernel));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *output_ptr = acc_output.ptr(rect_output.lo);
  //float *replica_ptr = acc_output.ptr(rect_replica.lo);
  const float *kernel_ptr = acc_kernel.ptr(rect_kernel.lo);
  const float *bias_ptr = acc_bias.ptr(rect_bias.lo);
  float *pre_relu_ptr = (m->relu) ? m->pre_relu : output_ptr;

  //checkCUDA(hipMemcpy(replica_ptr, input_ptr, rect_input.volume() * sizeof(float),
  //                     hipMemcpyDeviceToDevice));
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        output_channels, batch_size, input_channels,
                        &alpha, kernel_ptr, input_channels,
                        input_ptr, input_channels, &beta,
                        output_ptr, output_channels));
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        output_channels, batch_size, 1,
                        &alpha, bias_ptr, 1,
                        one_ptr, 1, &alpha,
                        pre_relu_ptr, output_channels));
  if (m->relu) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
                                      &alpha, m->outputTensor, pre_relu_ptr,
                                      &beta, m->outputTensor, output_ptr));
  }
}

void Linear::forward(const CnnModel& model)
{
  ArgumentMap argmap;
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, model.fc_part_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(LINEAR_FWD_TASK_ID, model.fc_part_is,
                         TaskArgument(NULL, 0), argmap);
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(output.partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  launcher.add_field(1, FID_DATA);
  //launcher.add_region_requirement(
  //    RegionRequirement(locals[0].partition, 0/*projection id*/,
  //                      WRITE_DISCARD, EXCLUSIVE, locals[0].region));
  //launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(locals[1].partition, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[1].region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(locals[2].partition, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[2].region));
  launcher.add_field(3, FID_DATA);

  runtime->execute_index_space(ctx, launcher);
}

/*
  regions[0](O): replica
  regions[1](I): output
  regions[2](I/O): filter
  regions[3](I/O): bias
*/
__host__
void Linear::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  float alpha = 1.0f, beta = 0.0f;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  int input_channels = m->input_channels;
  int output_channels = m->output_channels;
  int batch_size = m->batch_size;
  const float *one_ptr = m->one_ptr;
  const AccessorWO<float, 2> acc_replica(regions[0], FID_DATA);
  const AccessorRO<float, 2> acc_output(regions[1], FID_DATA);
  const AccessorRW<float, 2> acc_kernel(regions[2], FID_DATA);
  const AccessorRW<float, 2> acc_bias(regions[3], FID_DATA);
  Rect<2> rect_output, rect_replica, rect_kernel, rect_bias;
  rect_replica = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_kernel = runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  // make sure the sizes match
  assert(rect_replica.volume() == input_channels * batch_size);
  assert(rect_output.volume() == output_channels * batch_size);
  assert(rect_kernel.volume() == input_channels * output_channels);
  assert(rect_bias.volume() == output_channels);
  assert(acc_replica.accessor.is_dense_arbitrary(rect_replica));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  assert(acc_kernel.accessor.is_dense_arbitrary(rect_kernel));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  float *replica_ptr = acc_replica.ptr(rect_replica.lo);
  const float *output_ptr = acc_output.ptr(rect_output.lo);
  float *kernel_ptr = acc_kernel.ptr(rect_kernel.lo);
  float *bias_ptr = acc_bias.ptr(rect_bias.lo);

  //if (m->relu) {
  //  int n = rect_output.volume();
  //  reluBackward<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(output_ptr, m->pre_relu, n);
  //}

  // Compute weight gradiant
}

/*
  regions[0](O): input
  regions[1..fc_num_par_c]: subreplicas
*/
__host__
void Linear::backward2_task(const Task *task,
                            const std::vector<PhysicalRegion> &regions,
                            Context ctx, Runtime *runtime)
{
  float alpha = 1.0f, beta = 0.0f;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  int input_channels = m->input_channels;
  int output_channels = m->output_channels;
  int batch_size = m->batch_size;
  const AccessorWO<float, 2> acc_input(regions[0], FID_DATA);
  Rect<2> rect_input, rect_replica;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  float *input_ptr = acc_input.ptr(rect_input.lo);
  for (int i = 1; i < task->regions.size(); i++) {
    const AccessorRO<float, 2> acc_replica(regions[i], FID_DATA);
    rect_replica = runtime->get_index_space_domain(ctx, task->regions[i].region.get_index_space());
    assert(rect_replica.volume() == rect_input.volume());
    assert(acc_replica.accessor.is_dense_arbitrary(rect_replica));
    const float *replica_ptr = acc_replica.ptr(rect_replica.lo);
    if (i == 1)
      checkCUDA(hipblasScopy(m->handle.blas, rect_input.volume(),
                            replica_ptr, 1, input_ptr, 1));
    else
      checkCUDA(hipblasSaxpy(m->handle.blas, rect_input.volume(),
                            &alpha, replica_ptr, 1, input_ptr, 1));
  }
}

void Linear::backward(const CnnModel& model)
{
  ArgumentMap argmap;
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, model.fc_part_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  {
    IndexLauncher launcher(LINEAR_BWD_TASK_ID, model.fc_part_is,
                           TaskArgument(NULL, 0), argmap);
    //launcher.add_region_requirement(
    //    RegionRequirement(inputs[0].partition, 0/*projection id*/,
    //                      READ_ONLY, EXCLUSIVE, inputs[0].region));
    //launcher.add_field(0, FID_DATA);
    // Replica tensor
    launcher.add_region_requirement(
        RegionRequirement(locals[0].partition, 0/*projection id*/,
                          WRITE_DISCARD, EXCLUSIVE, locals[0].region));
    launcher.add_field(0, FID_DATA);
    launcher.add_region_requirement(
        RegionRequirement(output.partition, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, output.region));
    launcher.add_field(1, FID_DATA);
    // Filter tensor
    launcher.add_region_requirement(
        RegionRequirement(locals[1].partition, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, locals[1].region));
    launcher.add_field(2, FID_DATA);
    // Bias tensor
    launcher.add_region_requirement(
        RegionRequirement(locals[2].partition, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, locals[2].region));
    launcher.add_field(3, FID_DATA);
    runtime->execute_index_space(ctx, launcher);
  }
  {
    // We aggregate parameters from replica tensor to input tensor
    IndexLauncher launcher2(LINEAR_BWD2_TASK_ID, model.fc_part_is,
                            TaskArgument(NULL, 0), argmap);
    launcher2.add_region_requirement(
        RegionRequirement(inputs[0].partition, 0/*projection id*/,
                          WRITE_DISCARD, EXCLUSIVE, inputs[0].region));
    launcher2.add_field(0, FID_DATA);
    for (int i = 0; i < model.config.fc_num_par_c; i++) {
      launcher2.add_region_requirement(
          RegionRequirement(replica_sub_lps[i], 0/*partition id*/,
                            READ_ONLY, EXCLUSIVE, locals[0].region));
      launcher2.add_field(i + 1, FID_DATA);
    }
    runtime->execute_index_space(ctx, launcher2);
  }
}

