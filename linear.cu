#include "hip/hip_runtime.h"
/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ops.h"
#include "cnn_helper.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hiprand.h>

Tensor CnnModel::add_linear_layer(Tensor input, int output_channels, bool relu)
{
  assert(input.numDim == 2);
  Linear *li = new Linear(config, input, fc_part_is, output_channels, relu);
  layers.push_back(li);
  return li->output;
}

Linear::Linear(CnnConfig config, Tensor input, IndexSpaceT<2> part_is,
               int output_channels, bool _relu)
: Op(input), relu(_relu)
{
  assert(input.numDim == 2);
  Context ctx = config.lg_ctx;
  HighLevelRuntime* runtime = config.lg_hlr;

  FieldSpace fs = runtime->create_field_space(ctx);
  {
    FieldAllocator allocator = runtime->create_field_allocator(ctx, fs);
    allocator.allocate_field(sizeof(float), FID_DATA);
  }

  Rect<2, coord_t> output_rect(Point<2>(0, 0), Point<2>(output_channels-1, input.adim[1]-1));
  IndexSpaceT<2> output_is = runtime->create_index_space(ctx, output_rect);
  LogicalRegion output_lr = runtime->create_logical_region(ctx, output_is, fs);
  Transform<2, 2, coord_t> transform;
  int extent_c = (output_channels + config.fc_num_par_c - 1) / config.fc_num_par_c;
  int extent_n = (input.adim[3] + config.fc_num_par_n - 1) / config.fc_num_par_n;
  Rect<2, coord_t> extent(Point<2>(0, 0), Point<2>(extent_c-1, extent_n-1));
  transform[0][0] = extent_c; transform[0][1] = 0;
  transform[1][0] = 0; transform[1][1] = extent_n;
  IndexPartition output_ip =
    runtime->create_partition_by_restriction(ctx, output_is, part_is, transform, extent);
  LogicalPartition output_lp = runtime->get_logical_partition(ctx, output_lr, output_ip);
  
  int input_channels = input.adim[0];
  Rect<2, coord_t> kernel_rect(Point<2>(0, 0), Point<2>(output_channels * input_channels-1, config.fc_num_par_n-1));
  IndexSpaceT<2> kernel_is = runtime->create_index_space(ctx, kernel_rect);
  LogicalRegion kernel_lr = runtime->create_logical_region(ctx, kernel_is, fs);
  transform[0][0] = extent_c * input_channels;
  transform[1][1] = 1;
  Rect<2, coord_t> extent_k(Point<2>(0, 0), Point<2>(extent_c*input_channels-1, 0));
  printf("extent_k(%dx%d %d)\n", extent_c, input_channels, 1);
  IndexPartition kernel_ip =
    runtime->create_partition_by_restriction(ctx, kernel_is, part_is, transform, extent_k);
  LogicalPartition kernel_lp = runtime->get_logical_partition(ctx, kernel_lr, kernel_ip);
  TensorWithGrad kernel_tensor;
  kernel_tensor.region = kernel_lr;
  kernel_tensor.partition = kernel_lp;
  locals[0] = kernel_tensor;

  Rect<2, coord_t> bias_rect(Point<2>(0, 0), Point<2>(output_channels-1, config.fc_num_par_n-1));
  IndexSpaceT<2> bias_is = runtime->create_index_space(ctx, bias_rect);
  LogicalRegion bias_lr = runtime->create_logical_region(ctx, bias_is, fs);
  transform[0][0] = extent_c;
  transform[1][1] = 1;
  Rect<2, coord_t> extent_b(Point<2>(0, 0), Point<2>(extent_c-1,0));
  IndexPartition bias_ip =
    runtime->create_partition_by_restriction(ctx, bias_is, part_is, transform, extent_b);
  LogicalPartition bias_lp = runtime->get_logical_partition(ctx, bias_lr, bias_ip);
  TensorWithGrad bias_tensor;
  bias_tensor.region = bias_lr;
  bias_tensor.partition = bias_lp;
  locals[1] = bias_tensor;

  output.numDim = 2;
  output.adim[0] = output_channels;
  output.adim[1] = input.adim[1];
  output.pdim[0] = extent_c;
  output.pdim[1] = extent_n;
  output.region = output_lr;
  output.partition = output_lp;

  // Every partition reads all input_channels
  transform[0][0] = 0;
  transform[1][1] = extent_n;
  Rect<2, coord_t> extent_i(Point<2>(0, 0), Point<2>(input_channels-1, extent_n-1));
  IndexSpaceT<2> input_is = IndexSpaceT<2>(inputs[0].region.get_index_space());
  IndexPartition input_ip 
     = runtime->create_partition_by_restriction(ctx, input_is, part_is, transform, extent_i);
  input_lps[0] = runtime->get_logical_partition(ctx, inputs[0].region, input_ip);
}

/*
  regions[0]: output
  regions[1]: kernel
  regions[2]: bias
*/
OpMeta* Linear::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  const int BLKSIZE = 512;
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  const Linear* linear = (Linear*) task->args;
  CnnHandle handle = *((const CnnHandle*) task->local_args);
  const AccessorWO<float, 2> acc_kernel(regions[1], FID_DATA);
  const AccessorWO<float, 2> acc_bias(regions[2], FID_DATA);
  Rect<2> rect_kernel, rect_bias;
  rect_kernel = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  assert(acc_kernel.accessor.is_dense_arbitrary(rect_kernel));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  float* kernel_ptr = acc_kernel.ptr(rect_kernel.lo);
  float* bias_ptr = acc_bias.ptr(rect_bias.lo);
  hiprandGenerator_t genGPU;
  hiprandCreateGenerator(&genGPU, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(genGPU, 1234ULL);
  int input_channels = linear->inputs[0].adim[0];
  int output_channels = linear->output.pdim[0];
  int batch_size = linear->output.pdim[1];

  coord_t kernel_elements = input_channels * linear->output.pdim[0];
  float factor = 1.0f / sqrt(input_channels);
  assert(kernel_elements == rect_kernel.volume());
  hiprandGenerateUniform(genGPU, kernel_ptr, kernel_elements);
  int num_blocks = (kernel_elements + BLKSIZE - 1) / BLKSIZE;
  scale_kernel<<<num_blocks, BLKSIZE>>>(kernel_ptr, kernel_elements, -factor, factor);
  hiprandGenerateUniform(genGPU, bias_ptr, linear->output.pdim[0]);
  num_blocks = (linear->output.pdim[0] + BLKSIZE - 1) / BLKSIZE;
  scale_kernel<<<num_blocks, BLKSIZE>>>(bias_ptr, linear->output.pdim[0], -factor, factor);
  hiprandDestroyGenerator(genGPU);

  LinearMeta* m = new LinearMeta(handle);
  m->relu = linear->relu;
  m->input_channels = input_channels;
  m->output_channels = output_channels;
  m->batch_size = batch_size;
  float* dram_one_ptr = (float *) malloc(sizeof(float) * batch_size);
  for (int i = 0; i < batch_size; i++)
    dram_one_ptr[i] = 1.0f;
  checkCUDA(hipMalloc(&m->one_ptr, sizeof(float) * batch_size));
  checkCUDA(hipMemcpy(m->one_ptr, dram_one_ptr,
                       sizeof(float) * batch_size, hipMemcpyDeviceToDevice));
  if (m->relu) {
    checkCUDNN(hipdnnCreateActivationDescriptor(&m->actiDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, HIPDNN_ACTIVATION_RELU,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    checkCUDNN(hipdnnCreateTensorDescriptor(&m->outputTensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          batch_size, output_channels, 1, 1));
  }
  return m;
}

void Linear::init(const CnnModel& model)
{
  ArgumentMap argmap;
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, model.fc_part_is);
  int idx;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    CnnHandle handle = model.cnn_handlers[idx++];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(CnnHandle)));
  }
  IndexLauncher init_launcher(LINEAR_INIT_TASK_ID, model.fc_part_is,
                              TaskArgument(this, sizeof(Linear)), argmap);
  init_launcher.add_region_requirement(
      RegionRequirement(output.partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  init_launcher.add_field(0, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[0].partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, locals[0].region));
  init_launcher.add_field(1, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[1].partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, locals[1].region));
  init_launcher.add_field(2, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, init_launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*
  regions[0](I); input
  regions[1](O): output
  regions[2](I): kernel
  regions[3](I): bias
*/
__host__
void Linear::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  float alpha = 1.0f, beta = 0.0f;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  int input_channels = m->input_channels;
  int output_channels = m->output_channels;
  int batch_size = m->batch_size;
  const float *one_ptr = m->one_ptr;
  const AccessorRO<float, 2> acc_input(regions[0], FID_DATA);
  const AccessorWO<float, 2> acc_output(regions[1], FID_DATA);
  const AccessorRO<float, 2> acc_kernel(regions[2], FID_DATA);
  const AccessorRO<float, 2> acc_bias(regions[3], FID_DATA);
  Rect<2> rect_input, rect_output, rect_kernel, rect_bias;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_kernel = runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  // make sure the sizes match
  assert(rect_input.volume() == input_channels * batch_size);
  assert(rect_output.volume() == output_channels * batch_size);
  assert(rect_kernel.volume() == input_channels * output_channels);
  assert(rect_bias.volume() == output_channels);
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  assert(acc_kernel.accessor.is_dense_arbitrary(rect_kernel));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *output_ptr = acc_output.ptr(rect_output.lo);
  const float *kernel_ptr = acc_kernel.ptr(rect_kernel.lo);
  const float *bias_ptr = acc_bias.ptr(rect_bias.lo);

  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        output_channels, batch_size, input_channels,
                        &alpha, kernel_ptr, input_channels,
                        input_ptr, input_channels, &beta,
                        output_ptr, output_channels));
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        output_channels, batch_size, 1,
                        &alpha, bias_ptr, 1,
                        one_ptr, 1, &alpha,
                        output_ptr, output_channels));
  if (m->relu) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
                                      &alpha, m->outputTensor, output_ptr,
                                      &beta, m->outputTensor, output_ptr));
  }
}

void Linear::forward(const CnnModel& model)
{
  ArgumentMap argmap;
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, model.fc_part_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(LINEAR_FWD_TASK_ID, model.fc_part_is,
                         TaskArgument(NULL, 0), argmap);
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(output.partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(locals[0].partition, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[0].region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(locals[1].partition, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[1].region));
  launcher.add_field(3, FID_DATA);

  runtime->execute_index_space(ctx, launcher);
}

/*
  regions[0](I/O): input
  regions[1](I): output
  regions[2](I/O): filter
  regions[3](I/O): bias
*/
__host__
void Linear::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
}

void Linear::backward(const CnnModel& model)
{
}

