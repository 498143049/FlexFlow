/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "simulator.h"
#include "model.h"

Simulator::Simulator(FFHandler _handle, void* _base_ptr, size_t _capacity)
: handle(_handle), base_ptr((char*)_base_ptr), capacity(_capacity), offset(0),
warmup_times(5), repeat_times(10)
{
  hipEventCreate(&start_event);
  hipEventCreate(&end_event);
  conv2d_meta = new Conv2DMeta(handle);
  linear_meta = new LinearMeta(handle, 4096);
}

void Simulator::free_all()
{
  offset = 0;
}

void* Simulator::allocate(size_t num_elements, DataType type)
{
  size_t element_size = 0;
  switch (type) {
    case DT_FLOAT:
      element_size = sizeof(float);
      break;
    case DT_DOUBLE:
      element_size = sizeof(double);
      break;
    case DT_INT32:
      element_size = sizeof(int32_t);
      break;
    case DT_INT64:
      element_size = sizeof(int64_t);
      break;
    case DT_BOOLEAN:
      element_size = sizeof(bool);
      break;
    default:
      assert(false);
  }
  void* ret_ptr = base_ptr + offset;
  offset += element_size * num_elements;
  return ret_ptr;
}

