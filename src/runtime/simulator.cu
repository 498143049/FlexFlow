/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "simulator.h"
#include "model.h"

Simulator::Simulator(FFModel* model, void* _base_ptr, size_t _capacity)
: handle(_handle), base_ptr((char*)_base_ptr), capacity(_capacity), offset(0),
warmup_times(5), repeat_times(10)
{
  hipEventCreate(&start_event);
  hipEventCreate(&end_event);
  conv2d_meta = new Conv2DMeta(model->config.handle);
  linear_meta = new LinearMeta(model->config.handle, 4096);
  int num_nodes = model->config.numNodes;
  int gpus_per_node = model->config.workersPerNode;
  total_num_devices = num_nodes * gpus_per_node;
  // Create GPU compute device
  for (int i = 0; i < num_nodes; i++) 
    for (int j = 0; j < gpus_per_node; j++) {
      id_to_comp_device[i*gpus_per_node+j] = new Device(DEVICE_COMPUTE, i, j);
    }
  // Create inter GPU comm devices:
  for (int i = 0; i < total_num_devices; i++)
    for (int j = 0; j < total_num_devices; j++) {
      Device* src = id_to_comp_device[i];
      Device* dst = id_to_comp_device[j];
      if (src->node_id == dst->node_id && src != dst) {
        int hash = i * total_num_devices + j;
        ids_to_inter_gpu_comm_device[hash] = new Device(DEVICE_COMM,
            SimulatorConfig::inter_gpu_bandwidth);
      }
    }
  // Create gpu<->dram comm devices
  for (int i = 0; i < total_num_devices; i++) {
    id_to_gputodram_comm_device[i] = new Device(DEVICE_COMM,
        SimulatorConfig::gpu_dram_bandwidth);
    id_to_dramtogpu_comm_device[i] = new Device(DEVICE_COMM,
        SimulatorConfig::gpu_dram_bandwidth);
  }
  // Create inter node comm devices
  for (int i = 0; i < num_nodes; i++)
    for (int j = 0; j < num_nodes; j++)
      if (i != j) {
        int hash = i * total_num_devices + j;
        ids_to_inter_node_comm_device[hash] = new Device(DEVICE_COMM,
            SimulatorConfig::inter_node_bandwidth);
      }
  // Initialize task manager
  task_manager = new TaskManager(SimulatorConfig::max_num_tasks);
}
