#include "hip/hip_runtime.h"
/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::dense(std::string name,
                      const Tensor& input,
                      int outDim, 
                      ActiMode activation,
                      Initializer* kernel_initializer,
                      Initializer* bias_initializer)
{
  assert(input.numDim == 2);
  assert(config.strategies.find(name) != config.strategies.end());
  ParallelConfig pc = config.strategies[name];
  IndexSpaceT<2> task_is = IndexSpaceT<2>(get_or_create_task_is(pc));
  Linear *li = new Linear(name, config, input, task_is, outDim, activation,
                          kernel_initializer, bias_initializer);
  layers.push_back(li);
  return li->output;
}

//
// Deprecated -- TO BE REMOVED
//
Tensor FFModel::linear(std::string name,
                       const Tensor& input,
                       int out_dim,
                       ActiMode activation,
                       Initializer* kernel_initializer,
                       Initializer* bias_initializer)
{
  return dense(name, input, out_dim, activation,
               kernel_initializer, bias_initializer);
}

Linear::Linear(std::string _name,
               const FFConfig& _config,
               const Tensor& _input,
               const IndexSpaceT<2>& _task_is,
               int outDim,
               ActiMode _activation,
               Initializer* kernel_initializer,
               Initializer* bias_initializer)
: Op(_name, _input), task_is(_task_is),
  activation(_activation),
  profiling(_config.profiling)
  // in_channels(_input.adim[0]), out_channels(outDim)
{
  assert(_input.numDim == 2);
  Context ctx = _config.lg_ctx;
  HighLevelRuntime* runtime = _config.lg_hlr;
  Rect<2> part_rect = runtime->get_index_space_domain(ctx, task_is);
  fc_num_par_c = part_rect.hi[0] - part_rect.lo[0] + 1;
  int fc_num_par_n = part_rect.hi[1] - part_rect.lo[1] + 1;
  num_replica = fc_num_par_n;
  int inDim = _input.adim[0];

  printf("Linear fc_num_par_c(%d) fc_num_par_n(%d)\n", fc_num_par_c, fc_num_par_n);
  FieldSpace fs = _config.field_space;

  Rect<2> output_rect(Point<2>(0, 0), Point<2>(outDim-1, _input.adim[1]-1));
  IndexSpaceT<2> output_is = runtime->create_index_space(ctx, output_rect);
  LogicalRegion output_lr = runtime->create_logical_region(ctx, output_is, fs);
  LogicalRegion output_grad_lr = runtime->create_logical_region(ctx, output_is, fs);
  Transform<2, 2, coord_t> transform;
  int extent_c = (outDim + fc_num_par_c - 1) / fc_num_par_c;
  int extent_n = (_input.adim[1] + fc_num_par_n - 1) / fc_num_par_n;
  Rect<2> extent(Point<2>(0, 0), Point<2>(extent_c-1, extent_n-1));
  transform[0][0] = extent_c; transform[0][1] = 0;
  transform[1][0] = 0; transform[1][1] = extent_n;
  IndexPartition output_ip =
    runtime->create_partition_by_restriction(ctx, output_is, task_is, transform, extent);
  assert(runtime->is_index_partition_disjoint(ctx, output_ip));
  assert(runtime->is_index_partition_complete(ctx, output_ip));
  LogicalPartition output_lp = runtime->get_logical_partition(ctx, output_lr, output_ip);
  LogicalPartition output_grad_lp = runtime->get_logical_partition(ctx, output_grad_lr, output_ip);

  // Note: we only need replica's grad, so no need to create lr/lp for forward
  Rect<2, coord_t> replica_rect(Point<2>(0, 0),
                       Point<2>(inDim*fc_num_par_c-1, _input.adim[1]-1));
  IndexSpaceT<2> replica_is = runtime->create_index_space(ctx, replica_rect);
  LogicalRegion replica_lr = runtime->create_logical_region(ctx, replica_is, fs);
  transform[0][0] = inDim;
  transform[1][1] = extent_n;
  Rect<2, coord_t> extent_r(Point<2>(0, 0), Point<2>(inDim-1, extent_n-1));
  IndexPartition replica_ip =
    runtime->create_partition_by_restriction(ctx, replica_is, task_is, transform, extent_r);
  assert(runtime->is_index_partition_disjoint(ctx, replica_ip));
  assert(runtime->is_index_partition_complete(ctx, replica_ip));
  LogicalPartition replica_lp = runtime->get_logical_partition(ctx, replica_lr, replica_ip);
  Tensor replica_tensor;
  replica_tensor.region = LogicalRegion::NO_REGION;
  replica_tensor.part = LogicalPartition::NO_PART;
  replica_tensor.region_grad = replica_lr;
  replica_tensor.part_grad = replica_lp;
  locals[0] = replica_tensor;
  // Create subpartitions for backward prop aggregation
  for (int i = 0; i < fc_num_par_c; i++) {
    transform[0][0] = _input.pdim[0];
    transform[1][1] = _input.pdim[1];
    Rect<2, coord_t> ext(Point<2>(inDim*i, 0),
                         Point<2>(inDim*i + _input.pdim[0] - 1,
                                  _input.pdim[1]-1));
    IndexPartition ip =
      runtime->create_partition_by_restriction(ctx, replica_is, task_is, transform, ext);
    assert(runtime->is_index_partition_disjoint(ctx, ip));
    replica_sub_lps[i] = runtime->get_logical_partition(ctx, replica_lr, ip);
  }

  Rect<1, coord_t> kernel_rect(0, inDim * outDim - 1);
  Rect<2, coord_t> kernel_grad_rect(Point<2>(0, 0), Point<2>(outDim * inDim-1, fc_num_par_n-1));
  IndexSpaceT<1> kernel_is = runtime->create_index_space(ctx, kernel_rect);
  IndexSpaceT<2> kernel_grad_is = runtime->create_index_space(ctx, kernel_grad_rect);
  LogicalRegion kernel_lr = runtime->create_logical_region(ctx, kernel_is, fs);
  LogicalRegion kernel_grad_lr = runtime->create_logical_region(ctx, kernel_grad_is, fs);
  transform[0][0] = extent_c * inDim;
  transform[1][1] = 1;
  Rect<2, coord_t> extent_k_grad(Point<2>(0, 0), Point<2>(extent_c*inDim-1, 0));
  printf("extent_k(%dx%d %d)\n", extent_c, inDim, 1);
  IndexPartition kernel_grad_ip =
    runtime->create_partition_by_restriction(ctx, kernel_grad_is, task_is,
                                             transform, extent_k_grad);
  assert(runtime->is_index_partition_disjoint(ctx, kernel_grad_ip));
  assert(runtime->is_index_partition_complete(ctx, kernel_grad_ip));
  LogicalPartition kernel_grad_lp =
    runtime->get_logical_partition(ctx, kernel_grad_lr, kernel_grad_ip);
  Transform<1, 2, coord_t> trans;
  trans[0][0] = extent_c * inDim; trans[0][1] = 0;
  Rect<1, coord_t> extent_k(0, extent_c*inDim-1);
  IndexPartition kernel_ip =
    runtime->create_partition_by_restriction(ctx, kernel_is, task_is, trans, extent_k);
  LogicalPartition kernel_lp =
    runtime->get_logical_partition(ctx, kernel_lr, kernel_ip);
  Tensor kernel_tensor;
  kernel_tensor.region = kernel_lr;
  kernel_tensor.part = kernel_lp;
  kernel_tensor.region_grad = kernel_grad_lr;
  kernel_tensor.part_grad = kernel_grad_lp;
  locals[1] = kernel_tensor;
  kernel_initializer->init(ctx, runtime, &kernel_tensor);

  Rect<1, coord_t> bias_rect(0, outDim-1);
  Rect<2, coord_t> bias_grad_rect(Point<2>(0, 0), Point<2>(outDim-1, fc_num_par_n-1));
  IndexSpaceT<1> bias_is = runtime->create_index_space(ctx, bias_rect);
  IndexSpaceT<2> bias_grad_is = runtime->create_index_space(ctx, bias_grad_rect);
  LogicalRegion bias_lr = runtime->create_logical_region(ctx, bias_is, fs);
  LogicalRegion bias_grad_lr = runtime->create_logical_region(ctx, bias_grad_is, fs);
  transform[0][0] = extent_c;
  transform[1][1] = 1;
  Rect<2, coord_t> extent_b_grad(Point<2>(0, 0), Point<2>(extent_c-1,0));
  IndexPartition bias_grad_ip =
    runtime->create_partition_by_restriction(ctx, bias_grad_is, task_is,
                                             transform, extent_b_grad);
  assert(runtime->is_index_partition_disjoint(ctx, bias_grad_ip));
  assert(runtime->is_index_partition_complete(ctx, bias_grad_ip));
  LogicalPartition bias_grad_lp =
    runtime->get_logical_partition(ctx, bias_grad_lr, bias_grad_ip);
  trans[0][0] = extent_c; trans[0][1] = 0;
  Rect<1, coord_t> extent_b(0, extent_c-1);
  IndexPartition bias_ip =
    runtime->create_partition_by_restriction(ctx, bias_is, task_is, trans, extent_b);
  LogicalPartition bias_lp =
    runtime->get_logical_partition(ctx, bias_lr, bias_ip);
  Tensor bias_tensor;
  bias_tensor.region = bias_lr;
  bias_tensor.part = bias_lp;
  bias_tensor.region_grad = bias_grad_lr;
  bias_tensor.part_grad = bias_grad_lp;
  locals[2] = bias_tensor;
  bias_initializer->init(ctx, runtime, &bias_tensor);
  numLocals = 3;

  output.numDim = 2;
  output.adim[0] = outDim;
  output.adim[1] = _input.adim[1];
  output.pdim[0] = extent_c;
  output.pdim[1] = extent_n;
  output.region = output_lr;
  output.part = output_lp;
  output.region_grad = output_grad_lr;
  output.part_grad = output_grad_lp;

  // Every partition reads all inDim
  transform[0][0] = 0;
  transform[1][1] = extent_n;
  Rect<2, coord_t> extent_i(Point<2>(0, 0), Point<2>(inDim-1, extent_n-1));
  IndexSpaceT<2> input_is = IndexSpaceT<2>(inputs[0].region.get_index_space());
  IndexPartition input_ip 
     = runtime->create_partition_by_restriction(ctx, input_is, task_is, transform, extent_i);
  input_lps[0] = runtime->get_logical_partition(ctx, inputs[0].region, input_ip);
}

/*
  regions[0](I): input
  regions[1](O): output
  regions[2]: replica
  regions[3](I): kernel
  regions[4](I): bias
*/
OpMeta* Linear::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 5);
  assert(task->regions.size() == 5);
  const Linear* linear = (Linear*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  //const AccessorRO<float, 2> acc_input(regions[0], FID_DATA);
  //const AccessorWO<float, 2> acc_output(regions[1], FID_DATA);
  //const AccessorRO<float, 1> acc_kernel(regions[3], FID_DATA);
  //const AccessorRO<float, 1> acc_bias(regions[4], FID_DATA);
  Rect<2> rect_input, rect_output, rect_replica;
  Rect<1> rect_kernel, rect_bias;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_replica = runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_kernel = runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(ctx, task->regions[4].region.get_index_space());
  assert(rect_replica.volume() == rect_input.volume());
  //assert(acc_kernel.accessor.is_dense_arbitrary(rect_kernel));
  //assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  //const float* kernel_ptr = acc_kernel.ptr(rect_kernel.lo);
  //const float* bias_ptr = acc_bias.ptr(rect_bias.lo);
  int input_channels = rect_input.hi[0] - rect_input.lo[0] + 1;
  int output_channels = rect_output.hi[0] - rect_output.lo[0] + 1;
  int batch_size = linear->output.pdim[1];
  printf("init linear (input): in_c(%d) out_c(%d) batch_size(%d)\n", input_channels, output_channels, batch_size);
  LinearMeta* m = new LinearMeta(handle);
  m->activation = linear->activation;
  m->in_channels = input_channels;
  m->out_channels = output_channels;
  m->batch_size = batch_size;

  float* dram_one_ptr = (float *) malloc(sizeof(float) * batch_size);
  for (int i = 0; i < batch_size; i++)
    dram_one_ptr[i] = 1.0f;
  checkCUDA(hipMalloc(&m->one_ptr, sizeof(float) * batch_size));
  checkCUDA(hipMemcpy(m->one_ptr, dram_one_ptr,
                       sizeof(float) * batch_size, hipMemcpyHostToDevice));
  if (m->activation == AC_MODE_RELU) {
    checkCUDNN(hipdnnCreateActivationDescriptor(&m->actiDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, HIPDNN_ACTIVATION_RELU,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    checkCUDNN(hipdnnCreateTensorDescriptor(&m->outputTensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          batch_size, output_channels, 1, 1));
  }
  return m;
}

#ifdef DEADCODE
/*
  regions[0](O): filter
  regions[1](O): bias
*/
__host__
void Linear::init_para_task(const Task *task,
                            const std::vector<PhysicalRegion> &regions,
                            Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const Linear* linear = (Linear*) task->args;
  const AccessorWO<float, 1> acc_filter(regions[0], FID_DATA);
  const AccessorWO<float, 1> acc_bias(regions[1], FID_DATA);
  Rect<1> rect_filter, rect_bias;
  rect_filter = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  assert(acc_filter.accessor.is_dense_arbitrary(rect_filter));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  float *filter_ptr = acc_filter.ptr(rect_filter.lo);
  float *bias_ptr = acc_bias.ptr(rect_bias.lo);
  // init filter and bias
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  hiprandGenerator_t genGPU;
  hiprandCreateGenerator(&genGPU, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetStream(genGPU, stream);
  hiprandSetPseudoRandomGeneratorSeed(genGPU, 1234ULL);
  coord_t filter_elements = linear->in_channels * linear->out_channels;
  float factor = 1.0f / sqrt(linear->in_channels);
  assert(filter_elements == rect_filter.volume());
  hiprandGenerateUniform(genGPU, filter_ptr, filter_elements);
  scale_kernel<<<GET_BLOCKS(filter_elements), CUDA_NUM_THREADS>>>(
      filter_ptr, filter_elements, -factor, factor);
  hiprandGenerateUniform(genGPU, bias_ptr, linear->out_channels);
  assert(linear->out_channels == rect_bias.volume());
  scale_kernel<<<GET_BLOCKS(linear->out_channels), CUDA_NUM_THREADS>>>(
      bias_ptr, linear->out_channels, -factor, factor);
  hiprandDestroyGenerator(genGPU);
}
#endif

void Linear::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;

#ifdef DEADCODE
  // First we initialize the filter and bias parameters
  {
    TaskLauncher para_launcher(LINEAR_INIT_PARA_TASK_ID, TaskArgument(this, sizeof(Linear)));
    para_launcher.add_region_requirement(
        RegionRequirement(locals[1].region, WRITE_DISCARD, EXCLUSIVE, locals[1].region));
    para_launcher.add_field(0, FID_DATA);
    para_launcher.add_region_requirement(
        RegionRequirement(locals[2].region, WRITE_DISCARD, EXCLUSIVE, locals[2].region));
    para_launcher.add_field(1, FID_DATA);
    runtime->execute_task(ctx, para_launcher);
  }
#endif

  Rect<2> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[idx++];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  IndexLauncher init_launcher(LINEAR_INIT_TASK_ID, task_is,
                              TaskArgument(this, sizeof(Linear)), argmap);
  init_launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  init_launcher.add_field(0, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  init_launcher.add_field(1, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[0].part_grad, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, locals[0].region_grad));
  init_launcher.add_field(2, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[1].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[1].region));
  init_launcher.add_field(3, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[2].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[2].region));
  init_launcher.add_field(4, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, init_launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*
  regions[0](I); input
  regions[1](O): output
  regions[2](I): kernel
  regions[3](I): bias
*/
__host__
void Linear::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  float alpha = 1.0f, beta = 0.0f;
  const Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  int input_channels = m->in_channels;
  int output_channels = m->out_channels;
  int batch_size = m->batch_size;
  const float *one_ptr = m->one_ptr;
  const AccessorRO<float, 2> acc_input(regions[0], FID_DATA);
  const AccessorWO<float, 2> acc_output(regions[1], FID_DATA);
  const AccessorRO<float, 1> acc_kernel(regions[2], FID_DATA);
  const AccessorRO<float, 1> acc_bias(regions[3], FID_DATA);
  Rect<2> rect_input, rect_output;
  Rect<1> rect_kernel, rect_bias;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_kernel = runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  // make sure the sizes match
  assert(rect_input.volume() == input_channels * batch_size);
  assert(rect_output.volume() == output_channels * batch_size);
  assert(rect_kernel.volume() == input_channels * output_channels);
  assert(rect_bias.volume() == output_channels);
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  //assert(acc_replica.accessor.is_dense_arbitrary(rect_replica));
  assert(acc_kernel.accessor.is_dense_arbitrary(rect_kernel));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *output_ptr = acc_output.ptr(rect_output.lo);
  //float *replica_ptr = acc_output.ptr(rect_replica.lo);
  const float *kernel_ptr = acc_kernel.ptr(rect_kernel.lo);
  const float *bias_ptr = acc_bias.ptr(rect_bias.lo);
  //float *pre_relu_ptr = (m->relu) ? m->pre_relu : output_ptr;

  //checkCUDA(hipMemcpy(replica_ptr, input_ptr, rect_input.volume() * sizeof(float),
  //                     hipMemcpyDeviceToDevice));
  hipEvent_t t_start, t_end;
  if (linear->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        output_channels, batch_size, input_channels,
                        &alpha, kernel_ptr, input_channels,
                        input_ptr, input_channels, &beta,
                        output_ptr, output_channels));
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        output_channels, batch_size, 1,
                        &alpha, bias_ptr, 1,
                        one_ptr, 1, &alpha,
                        output_ptr, output_channels));
  if (m->activation != AC_MODE_NONE) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
                                      &alpha, m->outputTensor, output_ptr,
                                      &beta, m->outputTensor, output_ptr));
  }
  if (linear->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Linear forward time = %.2lfms\n", elapsed);
  }
}

void Linear::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(LINEAR_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Linear)), argmap);
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  launcher.add_field(1, FID_DATA);
  //launcher.add_region_requirement(
  //    RegionRequirement(locals[0].partition, 0/*projection id*/,
  //                      WRITE_DISCARD, EXCLUSIVE, locals[0].region));
  //launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(locals[1].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[1].region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(locals[2].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[2].region));
  launcher.add_field(3, FID_DATA);

  runtime->execute_index_space(ctx, launcher);
}

/*
  regions[0](I): input
  regions[1](O): replica_grad
  regions[2](I): output
  regions[3](I/O): output_grad
  regions[4](I): filter
  regions[5](O): filter_grad
  regions[6](O): bias_grad
*/
__host__
void Linear::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  assert(regions.size() == 7);
  assert(task->regions.size() == 7);
  float alpha = 1.0f, beta = 0.0f;
  const Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  int input_channels = m->in_channels;
  int output_channels = m->out_channels;
  int batch_size = m->batch_size;
  const float *one_ptr = m->one_ptr;
  const AccessorRO<float, 2> acc_input(regions[0], FID_DATA);
  const AccessorWO<float, 2> acc_replica_grad(regions[1], FID_DATA);
  const AccessorRO<float, 2> acc_output(regions[2], FID_DATA);
  const AccessorRW<float, 2> acc_output_grad(regions[3], FID_DATA);
  const AccessorRO<float, 1> acc_kernel(regions[4], FID_DATA);
  const AccessorWO<float, 2> acc_kernel_grad(regions[5], FID_DATA);
  const AccessorWO<float, 2> acc_bias_grad(regions[6], FID_DATA);
  Rect<2> rect_input, rect_replica_grad, rect_output, rect_output_grad,
          rect_kernel_grad, rect_bias_grad;
  Rect<1> rect_kernel;
  rect_input =
    runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_replica_grad =
    runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_output =
    runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_output_grad =
    runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  rect_kernel =
    runtime->get_index_space_domain(ctx, task->regions[4].region.get_index_space());
  rect_kernel_grad =
    runtime->get_index_space_domain(ctx, task->regions[5].region.get_index_space());
  rect_bias_grad =
    runtime->get_index_space_domain(ctx, task->regions[6].region.get_index_space());
  // make sure the sizes match
  assert(rect_input.volume() == input_channels * batch_size);
  assert(rect_replica_grad.volume() == input_channels * batch_size);
  assert(rect_output.volume() == output_channels * batch_size);
  assert(rect_output_grad.volume() == output_channels * batch_size);
  assert(rect_kernel.volume() == input_channels * output_channels);
  assert(rect_kernel_grad.volume() == input_channels * output_channels);
  assert(rect_bias_grad.volume() == output_channels);
  // make sure all regions are dense
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_replica_grad.accessor.is_dense_arbitrary(rect_replica_grad));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  assert(acc_output_grad.accessor.is_dense_arbitrary(rect_output_grad));
  assert(acc_kernel.accessor.is_dense_arbitrary(rect_kernel));
  assert(acc_kernel_grad.accessor.is_dense_arbitrary(rect_kernel_grad));
  assert(acc_bias_grad.accessor.is_dense_arbitrary(rect_bias_grad));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *replica_grad_ptr = acc_replica_grad.ptr(rect_replica_grad.lo);
  const float *output_ptr = acc_output.ptr(rect_output.lo);
  float *output_grad_ptr = acc_output_grad.ptr(rect_output_grad.lo);
  const float *kernel_ptr = acc_kernel.ptr(rect_kernel.lo);
  float *kernel_grad_ptr = acc_kernel_grad.ptr(rect_kernel_grad.lo);
  float *bias_grad_ptr = acc_bias_grad.ptr(rect_bias_grad.lo);

  hipEvent_t t_start, t_end;
  if (linear->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  if (m->activation == AC_MODE_RELU) {
    int n = rect_output.volume();
    reluBackward<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(output_grad_ptr, output_ptr, n);
  } else {
    // TODO: only support relu or none activation
    assert(m->activation == AC_MODE_NONE);
  }

  // Compute weight gradiant
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        input_channels, output_channels, batch_size,
                        &alpha, input_ptr, input_channels,
                        output_grad_ptr, output_channels,
                        &beta, kernel_grad_ptr, input_channels));
  // Compute bias gradiant
  checkCUDA(hipblasSgemv(m->handle.blas, HIPBLAS_OP_N,
                        output_channels, batch_size,
                        &alpha, output_grad_ptr, output_channels,
                        one_ptr, 1,
                        &beta, bias_grad_ptr, 1));
  // Compute data gradiant
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        input_channels, batch_size, output_channels,
                        &alpha, kernel_ptr, input_channels,
                        output_grad_ptr, output_channels,
                        &beta, replica_grad_ptr, input_channels));
  if (linear->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Linear backward time = %.2lfms\n", elapsed);
  }
}

/*
  regions[0](O): input_grad
  regions[1..fc_num_par_c]: subreplicas
*/
__host__
void Linear::backward2_task(const Task *task,
                            const std::vector<PhysicalRegion> &regions,
                            Context ctx, Runtime *runtime)
{
  float alpha = 1.0f;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  const AccessorWO<float, 2> acc_input(regions[0], FID_DATA);
  Rect<2> rect_input, rect_replica;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  float *input_ptr = acc_input.ptr(rect_input.lo);
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));

  for (int i = 1; i < task->regions.size(); i++) {
    const AccessorRO<float, 2> acc_replica(regions[i], FID_DATA);
    rect_replica = runtime->get_index_space_domain(ctx, task->regions[i].region.get_index_space());
    //printf("rect_replica.hi = %lld lo = %lld\n", rect_replica.hi[0], rect_replica.lo[0]);
    //printf("rect_replica.hi = %lld lo = %lld\n", rect_replica.hi[1], rect_replica.lo[1]);
    //printf("rect_input.hi = %lld lo = %lld\n", rect_input.hi[0], rect_input.lo[0]);
    //printf("rect_input.hi = %lld lo = %lld\n", rect_input.hi[1], rect_input.lo[1]);
    assert(rect_replica.volume() == rect_input.volume());
    assert(acc_replica.accessor.is_dense_arbitrary(rect_replica));
    const float *replica_ptr = acc_replica.ptr(rect_replica.lo);
    if (i == 1)
      checkCUDA(hipblasScopy(m->handle.blas, rect_input.volume(),
                            replica_ptr, 1, input_ptr, 1));
    else
      checkCUDA(hipblasSaxpy(m->handle.blas, rect_input.volume(),
                            &alpha, replica_ptr, 1, input_ptr, 1));
  }
}

void Linear::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  {
    IndexLauncher launcher(LINEAR_BWD_TASK_ID, task_is,
                           TaskArgument(this, sizeof(Linear)), argmap);
    // regions[0](I): input
    launcher.add_region_requirement(
        RegionRequirement(input_lps[0], 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, inputs[0].region));
    launcher.add_field(0, FID_DATA);
    // regions[1](O): replica_grad (we only need grad tensors)
    launcher.add_region_requirement(
        RegionRequirement(locals[0].part_grad, 0/*projection id*/,
                          WRITE_DISCARD, EXCLUSIVE, locals[0].region_grad));
    launcher.add_field(1, FID_DATA);
    // regions[2](I): output
    launcher.add_region_requirement(
        RegionRequirement(output.part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, output.region));
    launcher.add_field(2, FID_DATA);
    // regions[3](I/O): output_grad
    launcher.add_region_requirement(
        RegionRequirement(output.part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, output.region_grad));
    launcher.add_field(3, FID_DATA);
    // regions[4](I): filter
    launcher.add_region_requirement(
        RegionRequirement(locals[1].part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, locals[1].region));
    launcher.add_field(4, FID_DATA);
    // regions[5](O): filter_grad
    launcher.add_region_requirement(
        RegionRequirement(locals[1].part_grad, 0/*projection id*/,
                          WRITE_DISCARD, EXCLUSIVE, locals[1].region_grad));
    launcher.add_field(5, FID_DATA);
    // regions[6](O): bias_grad
    launcher.add_region_requirement(
        RegionRequirement(locals[2].part_grad, 0/*projection id*/,
                          WRITE_DISCARD, EXCLUSIVE, locals[2].region_grad));
    launcher.add_field(6, FID_DATA);
    runtime->execute_index_space(ctx, launcher);
  }
  {
    // We aggregate parameters from replica tensor to input tensor
    IndexLauncher launcher2(LINEAR_BWD2_TASK_ID, task_is,
                            TaskArgument(this, sizeof(Linear)), argmap);
    launcher2.add_region_requirement(
        RegionRequirement(inputs[0].part_grad, 0/*projection id*/,
                          WRITE_DISCARD, EXCLUSIVE, inputs[0].region_grad));
    launcher2.add_field(0, FID_DATA);
    for (int i = 0; i < fc_num_par_c; i++) {
      launcher2.add_region_requirement(
          RegionRequirement(replica_sub_lps[i], 0/*partition id*/,
                            READ_ONLY, EXCLUSIVE, locals[0].region_grad));
      launcher2.add_field(i + 1, FID_DATA);
    }
    runtime->execute_index_space(ctx, launcher2);
  }
}

#ifdef DEADCODE
/*
  regions[0](I/O): filter
  regions[1](I): filter_grad
  regions[2](I/O): bias
  regions[3](I): bias_grad
*/
__host__
void Linear::update_task(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  const Linear* linear = (Linear*) task->args;
  const AccessorRW<float, 1> acc_filter(regions[0], FID_DATA);
  const AccessorRO<float, 2> acc_filter_grad(regions[1], FID_DATA);
  const AccessorRW<float, 1> acc_bias(regions[2], FID_DATA);
  const AccessorRO<float, 2> acc_bias_grad(regions[3], FID_DATA);
  Rect<1> rect_filter, rect_bias;
  Rect<2> rect_filter_grad, rect_bias_grad;
  rect_filter =
    runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_filter_grad =
    runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_bias =
    runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_bias_grad =
    runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  size_t filter_size = rect_filter.volume();
  size_t bias_size = rect_bias.volume();
  assert(filter_size == linear->in_channels * linear->out_channels);
  assert(bias_size == linear->out_channels);
  printf("filter_size(%d) linear->num_replica(%d) rect_filter_grad(%d)\n", filter_size, linear->num_replica, rect_filter_grad.volume());
  assert(filter_size * linear->num_replica == rect_filter_grad.volume());
  assert(bias_size * linear->num_replica == rect_bias_grad.volume());
  assert(acc_filter.accessor.is_dense_arbitrary(rect_filter));
  assert(acc_filter_grad.accessor.is_dense_arbitrary(rect_filter_grad));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  assert(acc_bias_grad.accessor.is_dense_arbitrary(rect_bias_grad));
  float *filter_ptr = acc_filter.ptr(rect_filter.lo);
  const float *filter_grad_ptr = acc_filter_grad.ptr(rect_filter_grad.lo);
  float *bias_ptr = acc_bias.ptr(rect_bias.lo);
  const float *bias_grad_ptr = acc_bias_grad.ptr(rect_bias_grad.lo);
  updateGAS(filter_ptr, filter_grad_ptr, filter_size,
            linear->num_replica, linear->learning_rate);
  updateGAS(bias_ptr, bias_grad_ptr, bias_size,
            linear->num_replica, linear->learning_rate);
}

__host__
void Linear::update(const FFModel& ff)
{
  // Synchronize the learning rate
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  learning_rate = ff.config.learningRate;
  assert(num_replica > 0);
  // Only aggregate parameters if more than one replica
  if (num_replica > 1) {
    TaskLauncher launcher(LINEAR_UPD_TASK_ID, TaskArgument(this, sizeof(Linear)));
    launcher.add_region_requirement(
      RegionRequirement(locals[1].region, READ_WRITE, EXCLUSIVE, locals[1].region));
    launcher.add_field(0, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(locals[1].region_grad, READ_ONLY, EXCLUSIVE, locals[1].region_grad));
    launcher.add_field(1, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(locals[2].region, READ_WRITE, EXCLUSIVE, locals[2].region));
    launcher.add_field(2, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(locals[2].region_grad, READ_ONLY, EXCLUSIVE, locals[2].region_grad));
    launcher.add_field(3, FID_DATA);
    runtime->execute_task(ctx, launcher);
  }
}
#endif
