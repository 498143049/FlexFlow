/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ops.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hiprand.h>

Tensor CnnModel::add_pooling_layer(Tensor input,
                                   int kernel_h, int kernel_w,
                                   int stride_h, int stride_w,
                                   int padding_h, int padding_w, bool relu)
{
  assert(input.numDim == 4); /*NCHW*/
  Pooling2D *pool = new Pooling2D(config, input, part_is, kernel_h, kernel_w,
                                  stride_h, stride_w, padding_h, padding_w, relu);
  layers.push_back(pool);
  return pool->output;
}

Pooling2D::Pooling2D(CnnConfig config, Tensor input, IndexSpaceT<3> part_is,
                     int _kernel_h, int _kernel_w, int _stride_h, int _stride_w,
                     int _padding_h, int _padding_w, bool _relu)
: Op(input), kernel_h(_kernel_h), kernel_w(_kernel_w), stride_h(_stride_h),
  stride_w(_stride_w), padding_h(_padding_h), padding_w(_padding_w), relu(_relu)
{
  Context ctx = config.lg_ctx;
  HighLevelRuntime* runtime = config.lg_hlr;

  int input_w = input.adim[0];
  int input_h = input.adim[1];
  int output_w = 1 + (input_w + 2 * padding_w - kernel_w) / stride_w;
  int output_h = 1 + (input_h + 2 * padding_h - kernel_h) / stride_h;
  int output_nc = input.adim[3] * input.adim[2];
  FieldSpace fs = runtime->create_field_space(ctx);
  {
    FieldAllocator allocator = runtime->create_field_allocator(ctx, fs);
    allocator.allocate_field(sizeof(float), FID_DATA);
  }

  Realm::ZRect<3, coord_t> output_rect(Realm::ZPoint<3>(0, 0, 0),
                      Realm::ZPoint<3>(output_w-1, output_h-1, output_nc-1));
  IndexSpaceT<3> output_is = runtime->create_index_space(ctx, output_rect);
  LogicalRegion output_lr = runtime->create_logical_region(ctx, output_is, fs);
  Realm::ZMatrix<3, 3, coord_t> transform;
  int extent_w = (output_w + config.num_par_w - 1) / config.num_par_w;
  int extent_h = (output_h + config.num_par_h - 1) / config.num_par_h;
  int extent_nc = output_nc / config.num_par_n;
  assert(output_nc % config.num_par_n == 0);
  Realm::ZRect<3, coord_t> extent(Realm::ZPoint<3>(0, 0, 0),
                      Realm::ZPoint<3>(extent_w-1, extent_h-1, extent_nc-1));
  transform[0][0] = extent_w; transform[0][1] = 0; transform[0][2] = 0;
  transform[1][0] = 0; transform[1][1] = extent_h; transform[1][2] = 0;
  transform[2][0] = 0; transform[2][1] = 0; transform[2][2] = extent_nc;
  IndexPartition output_ip =
    runtime->create_partition_by_restriction(ctx, output_is, part_is, transform, extent);
  LogicalPartition output_lp = runtime->get_logical_partition(ctx, output_lr, output_ip);

  output.numDim = 4;
  output.adim[0] = output_w;
  output.adim[1] = output_h;
  output.adim[2] = input.adim[2];
  output.adim[3] = input.adim[3];
  output.pdim[0] = extent_w;
  output.pdim[1] = extent_h;
  output.pdim[2] = output.adim[2];
  output.pdim[3] = output.adim[3];
  output.region = output_lr;
  output.partition = output_lp;

  // For now: the input lps are identical to inputs.partition
  input_lps[0] = inputs[0].partition;
}

/*
*/
OpMeta* Pooling2D::init_task(const Task *task,
                             const std::vector<PhysicalRegion> &regions,
                             Context ctx, Runtime *runtime)
{
  const Pooling2D* pool = (Pooling2D*) task->args;
  assert(regions.size() == 0);
  assert(regions.size() == 0);
  CnnHandle handle = *((const CnnHandle*) task->local_args);
  Pooling2DMeta* m = new Pooling2DMeta(handle);
  checkCUDNN(hipdnnCreateTensorDescriptor(&m->inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&m->outputTensor));
  checkCUDNN(hipdnnCreatePoolingDescriptor(&m->poolDesc));

  printf("pool(inputDim): n(%d) c(%d) h(%d) w(%d)\n", pool->inputs[0].pdim[3],
        pool->inputs[0].pdim[2], pool->inputs[0].pdim[1], pool->inputs[0].pdim[0]);
  printf("pool(outputDim): n(%d) c(%d) h(%d) w(%d)\n", pool->output.pdim[3],
        pool->output.pdim[2], pool->output.pdim[1], pool->output.pdim[0]);
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        pool->inputs[0].pdim[3],
                                        pool->inputs[0].pdim[2],
                                        pool->inputs[0].pdim[1],
                                        pool->inputs[0].pdim[0]));

  checkCUDNN(hipdnnSetPooling2dDescriptor(m->poolDesc,
                                         HIPDNN_POOLING_MAX,
                                         HIPDNN_PROPAGATE_NAN,
                                         pool->kernel_h,
                                         pool->kernel_w,
                                         pool->padding_h,
                                         pool->padding_w,
                                         pool->stride_h,
                                         pool->stride_w));
  int n, c, h, w;
  checkCUDNN(hipdnnGetPooling2dForwardOutputDim(m->poolDesc,
                                               m->inputTensor,
                                               &n, &c, &h, &w));
  assert(n == pool->output.pdim[3]);
  assert(c == pool->output.pdim[2]);
  assert(h == pool->output.pdim[1]);
  assert(w == pool->output.pdim[0]);


  checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        n, c, h, w));
  return m;
}

void Pooling2D::init(const CnnModel& model)
{
  ArgumentMap argmap;
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Realm::ZRect<3> rect = runtime->get_index_space_domain(ctx, model.part_is);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    CnnHandle handle = model.cnn_handlers[idx++];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(CnnHandle)));
  }
  IndexLauncher init_launcher(POOL2D_INIT_TASK_ID, model.part_is,
                              TaskArgument(this, sizeof(Pooling2D)), argmap);
  idx = 0;
  FutureMap fm = runtime->execute_index_space(ctx, init_launcher);
  fm.wait_all_results();
  for (PointInRectIterator<3> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*
  regions[0](I): input
  regions[1](O): output
*/

void Pooling2D::forward_task(const Task *task,
                             const std::vector<PhysicalRegion> &regions,
                             Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  float alpha = 1.0f, beta = 0.0f;
  const Pooling2DMeta* m = *((Pooling2DMeta**) task->local_args);
  const FieldAccessor<READ_ONLY, float, 3> acc_input(regions[0], FID_DATA);
  const FieldAccessor<WRITE_DISCARD, float, 3> acc_output(regions[1], FID_DATA);
  Realm::ZRect<3> rect_input, rect_output;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *output_ptr = acc_output.ptr(rect_output.lo);

  checkCUDNN(hipdnnPoolingForward(m->handle.dnn, m->poolDesc,
                                 &alpha, m->inputTensor, input_ptr,
                                 &beta, m->outputTensor, output_ptr));
}

void Pooling2D::forward(const CnnModel& model)
{
  ArgumentMap argmap;
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Realm::ZRect<3> rect = runtime->get_index_space_domain(ctx, model.part_is);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    printf("mp.pointer = %llx\n", mp);
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(POOL2D_FWD_TASK_ID, model.part_is,
                         TaskArgument(NULL, 0), argmap);
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(output.partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  launcher.add_field(1, FID_DATA);

  runtime->execute_index_space(ctx, launcher);
}

void Pooling2D::backward_task(const Task *task,
                              const std::vector<PhysicalRegion> &regions,
                              Context ctx, Runtime *runtime)
{
}

void Pooling2D::backward(const CnnModel& model)
{
}
