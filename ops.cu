#include "hip/hip_runtime.h"
/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "ops.h"

CnnHandle init_cudnn(const Task *task,
                     const std::vector<PhysicalRegion> &regions,
                     Context ctx, HighLevelRuntime *runtime)
{
  assert(regions.size() == 0);
  assert(task->arglen == sizeof(size_t));
  size_t workSpaceSize = *(const size_t*) task->args;
  CnnHandle handle;
  handle.workSpaceSize = workSpaceSize;
  printf("workSpaceSize = %zu\n", workSpaceSize);
#ifndef DISABLE_COMPUTATION
  checkCUDA(hipblasCreate(&handle.blas));
  checkCUDNN(hipdnnCreate(&handle.dnn));
#endif
  checkCUDA(hipMalloc(&handle.workSpace, workSpaceSize));
  return handle;
}

Op::Op(Tensor input)
{
  inputs[0] = input;
}

CnnModel::CnnModel(int num_images, int height, int width,
                   int image_par, int height_par, int width_par,
                   int fc_par_n, int fc_par_c,
                   Context ctx, Runtime* runtime)
{
  config.lg_ctx = ctx;
  config.lg_hlr = runtime;
  config.num_par_w = width_par;
  config.num_par_h = height_par;
  config.num_par_n = image_par;
  config.num_workers = width_par * height_par * image_par;
  config.fc_num_par_c = fc_par_c;
  config.fc_num_par_n = fc_par_n;
  config.sm_num_par = fc_par_c * fc_par_n;
  Rect<3, coord_t> part_bounds(Point<3>(0, 0, 0), Point<3>(width_par-1, height_par-1, image_par-1));
  part_is = runtime->create_index_space(ctx, part_bounds);
  Rect<2, coord_t> fc_part_bounds(Point<2>(0, 0), Point<2>(fc_par_c-1, fc_par_n-1));
  fc_part_is = runtime->create_index_space(ctx, fc_part_bounds);
  Rect<1, coord_t> sm_part_bounds(Point<1>(0), Point<1>(config.sm_num_par-1));
  sm_part_is = runtime->create_index_space(ctx, sm_part_bounds);

  // input_images
  Rect<3, coord_t> image_rect(Point<3>(0, 0, 0), Point<3>(width-1, height-1, num_images*3-1));
  IndexSpaceT<3> image_is = runtime->create_index_space(ctx, image_rect);
  FieldSpace image_fs = runtime->create_field_space(ctx);
  {
    FieldAllocator allocator = runtime->create_field_allocator(ctx, image_fs);
    allocator.allocate_field(sizeof(float), FID_DATA);
  }
  LogicalRegion image_lr = runtime->create_logical_region(ctx, image_is, image_fs);
  LogicalRegion image_grad_lr = runtime->create_logical_region(ctx, image_is, image_fs);
  Transform<3, 3, coord_t> transform;
  int extent_w = width / width_par;
  int extent_h = height / height_par;
  int extent_nc = 3 * num_images / image_par;
  Rect<3, coord_t> extent(Point<3>(0, 0, 0), Point<3>(extent_w-1, extent_h-1, extent_nc-1));
  transform[0][0] = extent_w; transform[0][1] = 0; transform[0][2] = 0;
  transform[1][0] = 0; transform[1][1] = extent_h; transform[1][2] = 0;
  transform[2][0] = 0; transform[2][1] = 0; transform[2][2] = extent_nc;
  IndexPartition image_ip = 
    runtime->create_partition_by_restriction(ctx, image_is, part_is, transform, extent);
  LogicalPartition image_lp = runtime->get_logical_partition(ctx, image_lr, image_ip);
  LogicalPartition image_grad_lp =
    runtime->get_logical_partition(ctx, image_grad_lr, image_ip);
  input_image.numDim = 4;
  input_image.adim[0] = width;
  input_image.adim[1] = height;
  input_image.adim[2] = 3;
  input_image.adim[3] = num_images;
  input_image.pdim[0] = extent_w;
  input_image.pdim[1] = extent_h;
  input_image.pdim[2] = 3;
  input_image.pdim[3] = extent_nc / 3;
  input_image.region = image_lr;
  input_image.region_grad = image_grad_lr;
  input_image.partition = image_lp;
  input_image.partition_grad = image_grad_lp;

  // input_label
  Rect<1, coord_t> label_rect(Point<1>(0), Point<1>(num_images-1));
  IndexSpaceT<1> label_is = runtime->create_index_space(ctx, label_rect);
  FieldSpace label_fs = runtime->create_field_space(ctx);
  {
    FieldAllocator allocator = runtime->create_field_allocator(ctx, label_fs);
    allocator.allocate_field(sizeof(int), FID_DATA);
  }
  LogicalRegion label_lr = runtime->create_logical_region(ctx, label_is, label_fs);
  Transform<1, 1, coord_t> label_trans;
  int extent_n = (num_images + config.sm_num_par - 1) / config.sm_num_par;
  Rect<1, coord_t> label_extent(Point<1>(0), Point<1>(extent_n-1));
  label_trans[0][0] = extent_n;
  IndexPartition label_ip = runtime->create_partition_by_restriction(
                 ctx, label_is, sm_part_is, label_trans, label_extent);
  LogicalPartition label_lp = runtime->get_logical_partition(ctx, label_lr, label_ip);
  input_label.numDim = 1;
  input_label.adim[0] = num_images;
  input_label.pdim[0] = extent_n;
  input_label.region = label_lr;
  input_label.partition = label_lp;
};

void CnnModel::forward()
{
  for (size_t i = 0; i < layers.size(); i++) {
    layers[i]->forward(*this);
  }
}

void CnnModel::backward()
{
  int cm = 0;
  for (int i = layers.size() - 1; i >= 0; i--) {
    if (cm ++ == 6) break;
    layers[i]->backward(*this);
  }
}

__global__
void init_image_kernel(float* ptr, coord_t size)
{
  const coord_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < size) {
    ptr[tid] = 1.0f;
  }
}

__global__
void init_label_kernel(int* ptr, coord_t size)
{
  const coord_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < size) {
    ptr[tid] = 1;
  }
}

void CnnModel::init_images_task(const Task *task,
                                const std::vector<PhysicalRegion> &regions,
                                Context ctx, Runtime *runtime)
{
#ifndef DISABLE_COMPUTATION
  const int BLKSIZE = 512;
  const AccessorWO<float, 3> acc_image(regions[0], FID_DATA);
  Rect<3> rect_image;
  rect_image = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  assert(acc_image.accessor.is_dense_arbitrary(rect_image));
  float *image_ptr = acc_image.ptr(rect_image.lo);
  int num_blocks = (rect_image.volume() + BLKSIZE - 1) / BLKSIZE;
  init_image_kernel<<<num_blocks, BLKSIZE>>>(image_ptr, rect_image.volume());
#endif
}

void CnnModel::init_images()
{
  ArgumentMap argmap;
  Context ctx = config.lg_ctx;
  Runtime* runtime = config.lg_hlr;
  IndexLauncher launcher(IMAGE_INIT_TASK_ID, part_is,
                         TaskArgument(NULL, 0), argmap);
  launcher.add_region_requirement(
      RegionRequirement(input_image.partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, input_image.region));
  launcher.add_field(0, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

void CnnModel::init_labels_task(const Task *task,
                                const std::vector<PhysicalRegion> &regions,
                                Context ctx, Runtime *runtime)
{
  const int BLKSIZE = 512;
  const AccessorWO<int, 1> acc_label(regions[0], FID_DATA);
  Rect<1> rect_label;
  rect_label = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  assert(acc_label.accessor.is_dense_arbitrary(rect_label));
  int *label_ptr = acc_label.ptr(rect_label.lo);
  int num_blocks = (rect_label.volume() + BLKSIZE - 1) / BLKSIZE;
  init_label_kernel<<<num_blocks, BLKSIZE>>>(label_ptr, rect_label.volume());
}

void CnnModel::init_labels()
{
  ArgumentMap argmap;
  Context ctx = config.lg_ctx;
  Runtime* runtime = config.lg_hlr;
  IndexLauncher launcher(LABEL_INIT_TASK_ID, sm_part_is,
                         TaskArgument(NULL, 0), argmap);
  launcher.add_region_requirement(
      RegionRequirement(input_label.partition, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, input_label.region));
  launcher.add_field(0, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  //fm.wait_all_results();
}

Tensor CnnModel::add_flat_layer(Tensor input)
{
  assert(input.numDim == 4);
  Flat *flat = new Flat(config, input, part_is, fc_part_is);
  layers.push_back(flat);
  return flat->output;
}

Flat::Flat(CnnConfig config, Tensor input,
           IndexSpaceT<3> part_is_3d,
           IndexSpaceT<2> part_is_2d)
: Op(input)
{
  Context ctx = config.lg_ctx;
  HighLevelRuntime* runtime = config.lg_hlr;
  FieldSpace fs = runtime->create_field_space(ctx);
  {
    FieldAllocator allocator = runtime->create_field_allocator(ctx, fs);
    allocator.allocate_field(sizeof(float), FID_DATA);
  }
  
  int output_c = input.adim[0] * input.adim[1] * input.adim[2];
  int output_n = input.adim[3];
  Rect<2, coord_t> output_rect(Point<2>(0, 0), Point<2>(output_c-1, output_n-1));
  IndexSpaceT<2> output_is = runtime->create_index_space(ctx, output_rect);
  LogicalRegion output_lr = runtime->create_logical_region(ctx, output_is, fs);
  LogicalRegion output_grad_lr =
    runtime->create_logical_region(ctx, output_is, fs);
  Transform<2, 2, coord_t> transform;
  int extent_c = input.pdim[0] * input.pdim[1] * input.pdim[2];
  int extent_n = input.pdim[3];
  Rect<2, coord_t> extent(Point<2>(0, 0), Point<2>(extent_c-1,extent_n-1));
  transform[0][0] = extent_c; transform[0][1] = 0;
  transform[1][0] = 0; transform[1][1] = extent_n;
  IndexPartition output_ip =
    runtime->create_partition_by_restriction(ctx, output_is, part_is_2d, transform, extent);
  assert(runtime->is_index_partition_disjoint(ctx, output_ip));
  assert(runtime->is_index_partition_complete(ctx, output_ip));
  LogicalPartition output_lp = runtime->get_logical_partition(ctx, output_lr, output_ip);
  LogicalPartition output_grad_lp =
    runtime->get_logical_partition(ctx, output_grad_lr, output_ip);
  output.numDim = 2;
  output.adim[0] = output_c;
  output.adim[1] = output_n;
  output.pdim[0] = extent_c;
  output.pdim[1] = extent_n;
  output.region = output_lr;
  output.region_grad = output_lr;
  output.partition = output_lp;
  output.partition_grad = output_grad_lp;
  printf("Create flat layer: input(N=%d C=%d H=%d W=%d) -> output(N=%d C=%d)\n",
         input.adim[3], input.adim[2], input.adim[1], input.adim[0], output.adim[1], output.adim[0]);
 
  FieldSpace proj_fs = runtime->create_field_space(ctx);
  {
    FieldAllocator allocator = runtime->create_field_allocator(ctx, proj_fs);
    allocator.allocate_field(sizeof(Rect<2>), FID_DATA);
  }
  LogicalRegion proj_lr = runtime->create_logical_region(ctx, part_is_3d, proj_fs);
  InlineLauncher launcher(RegionRequirement(proj_lr, WRITE_DISCARD, EXCLUSIVE, proj_lr)
                                           .add_field(FID_DATA));
  PhysicalRegion proj_pr = runtime->map_region(ctx, launcher);
  proj_pr.wait_until_valid();
  coord_t subtotal = 0;
  {
    const FieldAccessor<WRITE_DISCARD, Rect<2>, 3, coord_t,
              Realm::AffineAccessor<Rect<2>, 3, coord_t> > ra(proj_pr, FID_DATA);
    Rect<3> rect = runtime->get_index_space_domain(ctx, part_is_3d);
    for(PointInRectIterator<3> pir(rect); pir(); ++pir) {
      IndexSpace subspace = runtime->get_index_subspace(input.partition.get_index_partition(), *pir);
      Rect<3> subrect = runtime->get_index_space_domain(ctx, subspace);
      // Currently we assume the size of each subregion is divisible by output_n (i.e., batch size)
      assert(subrect.volume() % output_n == 0);
      coord_t subsize = subrect.volume() / output_n;
      ra[*pir] = Rect<2>(Point<2>(subtotal, 0), Point<2>(subtotal + subsize - 1, output_n - 1));
      subtotal += subsize;
    }
  }
  runtime->unmap_region(ctx, proj_pr);
  Transform<3, 3, coord_t> proj_trans;
  proj_trans[0][0] = 1; proj_trans[0][1] = 0; proj_trans[0][2] = 0;
  proj_trans[1][0] = 0; proj_trans[1][1] = 1; proj_trans[1][2] = 0;
  proj_trans[2][0] = 0; proj_trans[2][1] = 0; proj_trans[2][2] = 1;
  Rect<3, coord_t> proj_extent(Point<3>(0, 0, 0), Point<3>(0, 0, 0));
  IndexPartition proj_ip =
    runtime->create_partition_by_restriction(ctx, part_is_3d, part_is_3d, proj_trans, proj_extent);
  LogicalPartition proj_lp = runtime->get_logical_partition(ctx, proj_lr, proj_ip);
  IndexPartition flat_ip =
    runtime->create_partition_by_image_range(ctx, output_is,
                         proj_lp, proj_lr, FID_DATA, part_is_3d);
  assert(runtime->is_index_partition_disjoint(ctx, flat_ip));
  assert(runtime->is_index_partition_complete(ctx, flat_ip));
  flat_lp = runtime->get_logical_partition(ctx, output_lr, flat_ip);
  flat_grad_lp = runtime->get_logical_partition(ctx, output_grad_lr, flat_ip);
  return;
/*
  Transform<2, 3, coord_t> flat_trans;
  flat_trans[0][0] = input.pdim[0] * input.pdim[1] * input.adim[2];
  flat_trans[0][1] = input.adim[0] * input.pdim[1] * input.adim[2];
  flat_trans[0][2] = 0;
  flat_trans[1][0] = 0;
  flat_trans[1][1] = 0;
  flat_trans[1][2] = input.pdim[3];
  IndexPartition flat_ip =
    runtime->create_partition_by_restriction(ctx, output_is, part_is_3d, flat_trans, extent);
  flat_lp = runtime->get_logical_partition(ctx, output_lr, flat_ip);
*/
}

OpMeta* Flat::init_task(const Task *task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime *runtime)
{
  CnnHandle handle = *((const CnnHandle*) task->local_args);
  FlatMeta* m = new FlatMeta(handle);
  return m;
}

void Flat::init(const CnnModel& model)
{
  ArgumentMap argmap;
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<3> rect = runtime->get_index_space_domain(ctx, model.part_is);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    CnnHandle handle = model.cnn_handlers[idx++];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(CnnHandle)));
  }

  IndexLauncher init_launcher(FLAT_INIT_TASK_ID, model.part_is,
                              TaskArgument(this, sizeof(Flat)), argmap);
  FutureMap fm = runtime->execute_index_space(ctx, init_launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*
  regions[0](I): input
  regions[1](I): output
*/  
void Flat::forward_task(const Task *task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime *runtime)
{
#ifndef DISABLE_COMPUTATION
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const AccessorRO<float, 3> acc_input(regions[0], FID_DATA);
  const AccessorWO<float, 2> acc_output(regions[1], FID_DATA);
  Rect<3> rect_input;
  Rect<2> rect_output;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  assert(rect_input.volume() == rect_output.volume());
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *output_ptr = acc_output.ptr(rect_output.lo);

  checkCUDA(hipMemcpy(output_ptr, input_ptr,
                       rect_input.volume() * sizeof(float),
                       hipMemcpyDeviceToDevice));
#endif
}

void Flat::forward(const CnnModel& model)
{
  ArgumentMap argmap;
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<3> rect = runtime->get_index_space_domain(ctx, model.part_is);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(FLAT_FWD_TASK_ID, model.part_is,
                         TaskArgument(NULL, 0), argmap);
  launcher.add_region_requirement(
      RegionRequirement(inputs[0].partition, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(flat_lp /*3D->2D partitions*/, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  launcher.add_field(1, FID_DATA);

  runtime->execute_index_space(ctx, launcher);
}

/*
  regions[0](O) : input_grad
  regions[1](I) : output_grad
*/
void Flat::backward_task(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
#ifndef DISABLE_COMPUTATION
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const AccessorWO<float, 3> acc_input_grad(regions[0], FID_DATA);
  const AccessorRO<float, 2> acc_output_grad(regions[1], FID_DATA);
  Rect<3> rect_input_grad;
  Rect<2> rect_output_grad;
  rect_input_grad =
    runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_output_grad =
    runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  assert(rect_input_grad.volume() == rect_output_grad.volume());
  assert(acc_input_grad.accessor.is_dense_arbitrary(rect_input_grad));
  assert(acc_output_grad.accessor.is_dense_arbitrary(rect_output_grad));
  float *input_grad_ptr = acc_input_grad.ptr(rect_input_grad.lo);
  const float *output_grad_ptr = acc_output_grad.ptr(rect_output_grad.lo);

  checkCUDA(hipMemcpy(input_grad_ptr, output_grad_ptr,
                       rect_input_grad.volume() * sizeof(float),
                       hipMemcpyDeviceToDevice));
#endif
}

void Flat::backward(const CnnModel& model)
{
  ArgumentMap argmap;
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<3> rect = runtime->get_index_space_domain(ctx, model.part_is);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(FLAT_BWD_TASK_ID, model.part_is,
                         TaskArgument(NULL, 0), argmap);
  launcher.add_region_requirement(
      RegionRequirement(inputs[0].partition_grad, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(flat_grad_lp /*3D->2D partitions*/, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, output.region_grad));
  launcher.add_field(1, FID_DATA);

  runtime->execute_index_space(ctx, launcher);
}
