/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "ops.h"

CnnHandle init_cudnn(const Task *task,
                     const std::vector<PhysicalRegion> &regions,
                     Context ctx, HighLevelRuntime *runtime)
{
  assert(regions.size() == 0);
  assert(task->arglen == sizeof(size_t));
  size_t workSpaceSize = *(const size_t*) task->args;
  CnnHandle handle;
  handle.workSpaceSize = workSpaceSize;
  checkCUDA(hipblasCreate(&handle.blas));
  checkCUDNN(hipdnnCreate(&handle.dnn));
  checkCUDA(hipMalloc(&handle.workSpace, workSpaceSize));
  return handle;
}

Op::Op(Tensor input)
{
  inputs[0] = input;
}

