#include "cuda_helper.h"
#include "flexflow_dataloader.h"

void ImgDataLoader::load_label(const Task *task,
                               const std::vector<PhysicalRegion> &regions,
                               Context ctx,
                               Runtime* runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  SampleIdxs* meta = (SampleIdxs*) task->local_args;
  TensorAccessorR<int, 2> acc_full_label(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<int, 2> acc_batch_label(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, false/*readOutput*/);
  int batch_size = acc_batch_label.rect.hi[1] - acc_batch_label.rect.lo[1] + 1;
  //FIXME: currently assume continous indices
  assert(batch_size == meta->num_samples);
  for (int i = 1; i < batch_size; i++)
    assert(meta->idxs[i] == meta->idxs[0] + i);
  const int* input_zc = acc_full_label.ptr + meta->idxs[0];
  copy_kernel<<<GET_BLOCKS(acc_batch_label.rect.volume()), CUDA_NUM_THREADS>>>(
    acc_batch_label.ptr, input_zc, acc_batch_label.rect.volume());
  checkCUDA(hipDeviceSynchronize());
}

void ImgDataLoader4D::load_input(const Task *task,
                                 const std::vector<PhysicalRegion> &regions,
                                 Context ctx,
                                 Runtime* runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  SampleIdxs* meta = (SampleIdxs*) task->local_args;
  TensorAccessorR<float, 4> acc_full_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 4> acc_batch_input(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, false/*readOutput*/);
  int batch_size = acc_batch_input.rect.hi[3] - acc_batch_input.rect.lo[3] + 1;
  int channels = acc_batch_input.rect.hi[2] - acc_batch_input.rect.lo[2] + 1;
  int height = acc_batch_input.rect.hi[1] - acc_batch_input.rect.lo[1] + 1;
  int width = acc_batch_input.rect.hi[0] - acc_batch_input.rect.lo[0] + 1;
  //FIXME: currently assume continous indices
  assert(batch_size == meta->num_samples);
  for (int i = 1; i < batch_size; i++)
    assert(meta->idxs[i] == meta->idxs[0] + i);
  const float* input_zc = acc_full_input.ptr + meta->idxs[0] * channels * height * width;
  //printf("load input %d %d %d %d\n", meta->idxs[0], channels, height, width);
  copy_kernel<<<GET_BLOCKS(acc_batch_input.rect.volume()), CUDA_NUM_THREADS>>>(
      acc_batch_input.ptr, input_zc, acc_batch_input.rect.volume());
  checkCUDA(hipDeviceSynchronize());
}

void ImgDataLoader2D::load_input(const Task *task,
                               const std::vector<PhysicalRegion> &regions,
                               Context ctx,
                               Runtime* runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  SampleIdxs* meta = (SampleIdxs*) task->local_args;
  TensorAccessorR<float, 2> acc_full_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> acc_batch_input(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, false/*readOutput*/);
  int batch_size = acc_batch_input.rect.hi[1] - acc_batch_input.rect.lo[1] + 1;
  int width = acc_batch_input.rect.hi[0] - acc_batch_input.rect.lo[0] + 1;
  //FIXME: currently assume continous indices
  assert(batch_size == meta->num_samples);
  for (int i = 1; i < batch_size; i++)
    assert(meta->idxs[i] == meta->idxs[0] + i);
  const float* input_zc = acc_full_input.ptr + meta->idxs[0] * width;
  //printf("load input %d %d %d %d\n", meta->idxs[0], channels, height, width);
  copy_kernel<<<GET_BLOCKS(acc_batch_input.rect.volume()), CUDA_NUM_THREADS>>>(
      acc_batch_input.ptr, input_zc, acc_batch_input.rect.volume());
  checkCUDA(hipDeviceSynchronize());
}

template<typename DT>
void SingleDataLoader::load_input_2d(const Task *task,
                                     const std::vector<PhysicalRegion> &regions,
                                     Context ctx,
                                     Runtime* runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  SampleIdxs* meta = (SampleIdxs*) task->local_args;
  TensorAccessorR<DT, 2> acc_full_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<DT, 2> acc_batch_input(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, false/*readOutput*/);
  int batch_size = acc_batch_input.rect.hi[1] - acc_batch_input.rect.lo[1] + 1;
  int width = acc_batch_input.rect.hi[0] - acc_batch_input.rect.lo[0] + 1;
  //FIXME: currently assume continous indices
  assert(batch_size == meta->num_samples);
  for (int i = 1; i < batch_size; i++)
    assert(meta->idxs[i] == meta->idxs[0] + i);
  const DT* input_zc = acc_full_input.ptr + meta->idxs[0] * width;
  //printf("load input %d %d %d %d\n", meta->idxs[0], channels, height, width);
  copy_kernel<DT><<<GET_BLOCKS(acc_batch_input.rect.volume()), CUDA_NUM_THREADS>>>(
      acc_batch_input.ptr, input_zc, acc_batch_input.rect.volume());
  checkCUDA(hipDeviceSynchronize());
}

template<typename DT>
void SingleDataLoader::load_input_4d(const Task *task,
                                     const std::vector<PhysicalRegion> &regions,
                                     Context ctx,
                                     Runtime* runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  SampleIdxs* meta = (SampleIdxs*) task->local_args;
  TensorAccessorR<DT, 4> acc_full_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<DT, 4> acc_batch_input(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, false/*readOutput*/);
  int batch_size = acc_batch_input.rect.hi[3] - acc_batch_input.rect.lo[3] + 1;
  int channels = acc_batch_input.rect.hi[2] - acc_batch_input.rect.lo[2] + 1;
  int height = acc_batch_input.rect.hi[1] - acc_batch_input.rect.lo[1] + 1;
  int width = acc_batch_input.rect.hi[0] - acc_batch_input.rect.lo[0] + 1;
  //FIXME: currently assume continous indices
  assert(batch_size == meta->num_samples);
  for (int i = 1; i < batch_size; i++)
    assert(meta->idxs[i] == meta->idxs[0] + i);
  const DT* input_zc = acc_full_input.ptr + meta->idxs[0] * channels * height * width;
  //printf("load input %d %d %d %d\n", meta->idxs[0], channels, height, width);
  copy_kernel<DT><<<GET_BLOCKS(acc_batch_input.rect.volume()), CUDA_NUM_THREADS>>>(
      acc_batch_input.ptr, input_zc, acc_batch_input.rect.volume());
  checkCUDA(hipDeviceSynchronize());
}

template void SingleDataLoader::load_input_4d<float>(const Task *task, const std::vector<PhysicalRegion> &regions, Context ctx, Runtime* runtime);
template void SingleDataLoader::load_input_2d<float>(const Task *task, const std::vector<PhysicalRegion> &regions, Context ctx, Runtime* runtime);
template void SingleDataLoader::load_input_2d<int>(const Task *task, const std::vector<PhysicalRegion> &regions, Context ctx, Runtime* runtime);